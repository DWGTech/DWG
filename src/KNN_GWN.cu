#include "hip/hip_runtime.h"
#include <vector>
#include <iomanip>
#include <random>
#include <queue>
#include <iostream>
#include <unordered_map>

#include "cukd/fcp.h"
#include "cukd/knn.h"
#include "cukd/builder.h"

#include "geometrycentral/utilities/vector3.h"

using namespace cukd;

float3* generatePoints(int N)
{
    //static int g_seed = 100000;
    //std::seed_seq seq{ g_seed++ };
    // std::random_device rd(seq());  // Will be used to obtain a seed for the random number engine
    //std::default_random_engine rd(seq);
    //std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    //std::uniform_int_distribution<> dist(0, N);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 100.0);

    std::cout << "generating " << N << " uniform random points" << std::endl;
    float3* d_points = 0;
    hipMallocManaged((char**)&d_points, N * sizeof(*d_points));
    if (!d_points)
        throw std::runtime_error("could not allocate points mem...");

    enum { num_dims = num_dims_of<float3>::value };
    for (int i = 0; i < N; i++)
        for (int d = 0; d < num_dims; d++) {
            ((float*)&d_points[i])[d] = (float)dis(gen);
        }
    return d_points;
}

template<typename CandidateList>
__global__
void d_knn(int* d_results,
    float3* d_queries,
    int      numQueries,
    const cukd::box_t<float3>* d_bounds,
    float3* d_nodes,
    int      numNodes,
    int      k,
    float    cutOffRadius)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= numQueries) return;

    CandidateList result(cutOffRadius);

    cct::knn(result,
        d_queries[tid], *d_bounds, d_nodes, numNodes);

    for (int i = 0; i < k; i++)
    {
        int c_index = tid * k + i;
        d_results[c_index] = result.get_pointID(i);
    }
}

extern "C" void GenerateKDTreeForGWN(std::vector<float>& points_pos_x, std::vector<float>& points_pos_y, std::vector<float>& points_pos_z,
    std::vector<float>& new_points_pos_x, std::vector<float>& new_points_pos_y, std::vector<float>& new_points_pos_z,
    std::vector<float>& bound_upper, std::vector<float>& bound_lower,
    bool out_index_map,
    std::unordered_map<geometrycentral::Vector3, int>& points_indices_map)
{
    using namespace cukd::common;

    /*=======================================================================================*/
    int numPoints = points_pos_x.size();
    float3* d_points = 0;
    hipMallocManaged((char**)&d_points, numPoints * sizeof(*d_points));
    if (!d_points)
        throw std::runtime_error("could not allocate points mem...");

    for (int i = 0; i < numPoints; i++) {
        d_points[i].x = points_pos_x[i];
        d_points[i].y = points_pos_y[i];
        d_points[i].z = points_pos_z[i];
    }
    /*=======================================================================================*/

    if (out_index_map)
    {
        for (int i = 0; i < numPoints; i++) {
            geometrycentral::Vector3 c_pos;
            c_pos.x = (float)d_points[i].x;
            c_pos.y = (float)d_points[i].y;
            c_pos.z = (float)d_points[i].z;

            points_indices_map.insert(std::pair<geometrycentral::Vector3, int>(c_pos, i));
        }
    }
    else
    {
        points_indices_map.clear();
    }

    /*=======================================================================================*/
    cukd::box_t<float3>* d_bounds;
    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<float3>));
    std::cout << "allocated memory for the world space bounding box ..." << std::endl;
    /*=======================================================================================*/

    /*=======================================================================================*/
    std::cout << "calling builder..." << std::endl;
    double t0 = getCurrentTime();
    cukd::buildTree(d_points, numPoints, d_bounds);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took "
        << prettyDouble(t1 - t0) << "s" << std::endl;
    /*=======================================================================================*/

    bound_upper[0] = d_bounds->upper.x; bound_upper[1] = d_bounds->upper.y; bound_upper[2] = d_bounds->upper.z;
    bound_lower[0] = d_bounds->lower.x; bound_lower[1] = d_bounds->lower.y; bound_lower[2] = d_bounds->lower.z;

    std::cout << "KDT Upper: (" << bound_upper[0] << " " << bound_upper[1] << " " << bound_upper[2] << ")" << std::endl;
    std::cout << "KDT Lower: (" << bound_lower[0] << " " << bound_lower[1] << " " << bound_lower[2] << ")" << std::endl;

    for (int i = 0; i < numPoints; i++)
    {
        new_points_pos_x[i] = (float)d_points[i].x;
        new_points_pos_y[i] = (float)d_points[i].y;
        new_points_pos_z[i] = (float)d_points[i].z;
    }

    hipFree(d_points);
    d_points = nullptr;

    hipFree(d_bounds);
    d_bounds = nullptr;
}

//void GenerateKDTreeForDWG(float3* d_points,
//    int points_size,
//    float3* bound_upper, float3* bound_lower,
//    std::unordered_map<geometrycentral::Vector3, int>& points_indices_map)
//{
//    using namespace cukd::common;
//
//    /*=======================================================================================*/
//    for (int i = 0; i < points_size; i++) {
//        geometrycentral::Vector3 c_pos;
//        c_pos.x = (double)d_points[i].x;
//        c_pos.y = (double)d_points[i].y;
//        c_pos.z = (double)d_points[i].z;
//
//        points_indices_map.insert(std::pair<geometrycentral::Vector3, int>(c_pos, i));
//    }
//    /*=======================================================================================*/
//
//    /*=======================================================================================*/
//    cukd::box_t<float3>* d_bounds;
//    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<float3>));
//    std::cout << "allocated memory for the world space bounding box ..." << std::endl;
//    /*=======================================================================================*/
//
//    /*=======================================================================================*/
//    std::cout << "calling builder..." << std::endl;
//    double t0 = getCurrentTime();
//    cukd::buildTree(d_points, points_size, d_bounds);
//    CUKD_CUDA_SYNC_CHECK();
//    double t1 = getCurrentTime();
//    std::cout << "done building tree, took "
//        << prettyDouble(t1 - t0) << "s" << std::endl;
//    /*=======================================================================================*/
//
//    bound_upper[0].x = d_bounds->upper.x; bound_upper[0].y = d_bounds->upper.y; bound_upper[0].z = d_bounds->upper.z;
//    bound_lower[0].x = d_bounds->lower.x; bound_lower[0].y = d_bounds->lower.y; bound_lower[0].z = d_bounds->lower.z;
//
//    std::cout << "KDT Upper: (" << bound_upper[0].x << " " << bound_upper[0].y << " " << bound_upper[0].z << ")" << std::endl;
//    std::cout << "KDT Lower: (" << bound_lower[0].x << " " << bound_lower[0].y << " " << bound_lower[0].z << ")" << std::endl;
//}

void GenerateKDTreeForDWG(
    float3* d_points,
    float3* d_normals,
    int points_size,
    float3* bound_upper, float3* bound_lower)
{
    using namespace cukd::common;

    /*=======================================================================================*/
    cukd::box_t<float3>* d_bounds;
    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<float3>));
    std::cout << "allocated memory for the world space bounding box ..." << std::endl;
    /*=======================================================================================*/

    /*=======================================================================================*/
    std::cout << "calling builder..." << std::endl;
    double t0 = getCurrentTime();
    cukd::buildDWGTree(d_points, d_normals, points_size, d_bounds);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took "
        << prettyDouble(t1 - t0) << "s" << std::endl;
    /*=======================================================================================*/

    bound_upper[0].x = d_bounds->upper.x; bound_upper[0].y = d_bounds->upper.y; bound_upper[0].z = d_bounds->upper.z;
    bound_lower[0].x = d_bounds->lower.x; bound_lower[0].y = d_bounds->lower.y; bound_lower[0].z = d_bounds->lower.z;

    std::cout << "KDT Upper: (" << bound_upper[0].x << " " << bound_upper[0].y << " " << bound_upper[0].z << ")" << std::endl;
    std::cout << "KDT Lower: (" << bound_lower[0].x << " " << bound_lower[0].y << " " << bound_lower[0].z << ")" << std::endl;
}

extern "C" void CudaKNNWithKDTreeforGWN(
    std::vector<float>& tree_pos_x, std::vector<float>& tree_pos_y, std::vector<float>& tree_pos_z,
    std::vector<float>& queries_pos_x, std::vector<float>& queries_pos_y, std::vector<float>& queries_pos_z,
    std::vector<float>& bound_upper, std::vector<float>& bound_lower,
    std::vector<std::vector<int>>& results_pos_index, 
    int k)
{
    using namespace cukd::common;

    float  cutOffRadius = std::numeric_limits<float>::infinity();

    /*=======================================================================================*/
    int numPoints = tree_pos_x.size();
    float3* d_points = 0;
    hipMallocManaged((char**)&d_points, numPoints * sizeof(*d_points));
    if (!d_points)
        throw std::runtime_error("could not allocate points mem...");

    for (int i = 0; i < numPoints; i++) {
        d_points[i].x = tree_pos_x[i];
        d_points[i].y = tree_pos_y[i];
        d_points[i].z = tree_pos_z[i];
    }
    /*=======================================================================================*/

    /*=======================================================================================*/
    cukd::box_t<float3>* d_bounds;
    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<float3>));
    d_bounds->lower.x = bound_lower[0]; d_bounds->lower.y = bound_lower[1]; d_bounds->lower.z = bound_lower[2];
    d_bounds->upper.x = bound_upper[0]; d_bounds->upper.y = bound_upper[1]; d_bounds->upper.z = bound_upper[2];
    /*=======================================================================================*/

    /*=======================================================================================*/
    double t0 = getCurrentTime();

    int numQueries = queries_pos_x.size();
    float3* d_queries = 0;
    hipMallocManaged((char**)&d_queries, numQueries * sizeof(*d_queries));
    if (!d_queries)
        throw std::runtime_error("could not allocate points mem...");

    for (int i = 0; i < numQueries; i++) {
        d_queries[i].x = queries_pos_x[i];
        d_queries[i].y = queries_pos_y[i];
        d_queries[i].z = queries_pos_z[i];
    }
    /*=======================================================================================*/

    int* d_results;
    CUKD_CUDA_CALL(MallocManaged((void**)&d_results, numQueries * k * sizeof(*d_results)));
    CUKD_CUDA_SYNC_CHECK();

    int bs = 128;
    int nb = divRoundUp(numQueries, bs);
    
    if (k == 108)
    {
        d_knn<HeapCandidateList<108>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 64)
    {
        d_knn<HeapCandidateList<64>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 36)
    {
        d_knn<HeapCandidateList<36>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 30)
    {
        d_knn<HeapCandidateList<30>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 20)
    {
        d_knn<FixedCandidateList<20>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 15)
    {
        d_knn<FixedCandidateList<15>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 10)
    {
        d_knn<FixedCandidateList<10>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 9)
    {
        d_knn<FixedCandidateList<9>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 8)
    {
        d_knn<FixedCandidateList<8>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 7)
    {
        d_knn<FixedCandidateList<7>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 6)
    {
        d_knn<FixedCandidateList<6>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 5)
    {
        d_knn<FixedCandidateList<5>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 4)
    {
        d_knn<FixedCandidateList<4>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 3)
    {
        d_knn<FixedCandidateList<3>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 2)
    {
        d_knn<FixedCandidateList<2>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 1)
    {
        d_knn<FixedCandidateList<1>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }


    CUKD_CUDA_SYNC_CHECK();

    double t1 = getCurrentTime();
    std::cout << "done searching, took "
        << prettyDouble(t1 - t0) << "s" << std::endl;

    for (int i = 0; i < numQueries; i++)
    {
        for (int u = 0; u < k; u++)
        {
            results_pos_index[i][u] = d_results[i * k + u];
        }
    }

    hipFree(d_points);
    d_points = nullptr;

    hipFree(d_bounds);
    d_bounds = nullptr;

    hipFree(d_queries);
    d_queries = nullptr;

    hipFree(d_results);
    d_results = nullptr;
}

void CudaKNNWithKDTreeforDWG(
    float3* d_points,
    int numPoints,
    float3* d_queries,
    int numQueries,
    float3* bound_upper, float3* bound_lower,
    int* d_results,
    int k)
{
    using namespace cukd::common;

    float  cutOffRadius = std::numeric_limits<float>::infinity();

    /*=======================================================================================*/
    cukd::box_t<float3>* d_bounds;
    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<float3>));
    d_bounds->lower.x = bound_lower[0].x; d_bounds->lower.y = bound_lower[0].y; d_bounds->lower.z = bound_lower[0].z;
    d_bounds->upper.x = bound_upper[0].x; d_bounds->upper.y = bound_upper[0].y; d_bounds->upper.z = bound_upper[0].z;
    /*=======================================================================================*/

    double t0 = getCurrentTime();

    int bs = 128;
    int nb = divRoundUp(numQueries, bs);

    //CUKD_CUDA_SYNC_CHECK();

    if (k == 108)
    {
        d_knn<HeapCandidateList<108>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 64)
    {
        d_knn<HeapCandidateList<64>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 36)
    {
        d_knn<HeapCandidateList<36>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 30)
    {
        d_knn<HeapCandidateList<30>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 20)
    {
        d_knn<HeapCandidateList<20>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 15)
    {
        d_knn<FixedCandidateList<15>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 10)
    {
        d_knn<FixedCandidateList<10>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 9)
    {
        d_knn<FixedCandidateList<9>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 8)
    {
        d_knn<FixedCandidateList<8>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 7)
    {
        d_knn<FixedCandidateList<7>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
     else if (k == 6)
    {
        d_knn<FixedCandidateList<6>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 5)
    {
        d_knn<FixedCandidateList<5>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 4)
    {
        d_knn<FixedCandidateList<4>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 3)
    {
        d_knn<FixedCandidateList<3>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 2)
    {
        d_knn<FixedCandidateList<2>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else if (k == 1)
    {
        d_knn<FixedCandidateList<1>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }

    CUKD_CUDA_SYNC_CHECK();

    double t1 = getCurrentTime();
    std::cout << "done searching, took "
        << prettyDouble(t1 - t0) << "s" << std::endl;
}

extern "C" void CudaKNNforGWN(std::vector<float>& points_pos_x, std::vector<float>& points_pos_y, std::vector<float>& points_pos_z,
                   std::vector<float>& queries_pos_x, std::vector<float>& queries_pos_y, std::vector<float>& queries_pos_z,
                   std::vector<std::vector<float>>& results_pos_x, std::vector<std::vector<float>>& results_pos_y, std::vector<std::vector<float>>& results_pos_z,
                   int k)
{
    using namespace cukd::common;

    float  cutOffRadius = std::numeric_limits<float>::infinity();

    /*=======================================================================================*/
    int numPoints = points_pos_x.size();
    float3* d_points = 0;
    hipMallocManaged((char**)&d_points, numPoints * sizeof(*d_points));
    if (!d_points)
        throw std::runtime_error("could not allocate points mem...");

    for (int i = 0; i < numPoints; i++) {
        d_points[i].x = points_pos_x[i];
        d_points[i].y = points_pos_y[i];
        d_points[i].z = points_pos_z[i];
    }
    /*=======================================================================================*/

    /*=======================================================================================*/
    cukd::box_t<float3>* d_bounds;
    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<float3>));
    std::cout << "allocated memory for the world space bounding box ..." << std::endl;
    /*=======================================================================================*/

    /*=======================================================================================*/
    std::cout << "calling builder..." << std::endl;
    double t0 = getCurrentTime();
    cukd::buildTree(d_points, numPoints, d_bounds);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took "
        << prettyDouble(t1 - t0) << "s" << std::endl;
    /*=======================================================================================*/

    /*=======================================================================================*/
    int numQueries = queries_pos_x.size();
    float3* d_queries = 0;
    hipMallocManaged((char**)&d_queries, numQueries * sizeof(*d_queries));
    if (!d_queries)
        throw std::runtime_error("could not allocate points mem...");

    for (int i = 0; i < numQueries; i++) {
        d_queries[i].x = queries_pos_x[i];
        d_queries[i].y = queries_pos_y[i];
        d_queries[i].z = queries_pos_z[i];
    }
    /*=======================================================================================*/

    int* d_results;
    CUKD_CUDA_CALL(MallocManaged((void**)&d_results, numQueries * k * sizeof(*d_results)));
    CUKD_CUDA_SYNC_CHECK();

    int bs = 128;
    int nb = divRoundUp(numQueries, bs);

    if (k == 30)
    {
        d_knn<HeapCandidateList<30>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }
    else
    {
        d_knn<FixedCandidateList<15>> << <nb, bs >> >
            (d_results,
                d_queries,
                numQueries,
                d_bounds,
                d_points,
                numPoints,
                k,
                cutOffRadius);
    }

    CUKD_CUDA_SYNC_CHECK();

    for (int i = 0; i < numQueries; i++)
    {
        for (int u = 0; u < k; u++)
        {
            results_pos_x[i][u] = (float)d_points[d_results[i * k + u]].x;
            results_pos_y[i][u] = (float)d_points[d_results[i * k + u]].y;
            results_pos_z[i][u] = (float)d_points[d_results[i * k + u]].z;
        }
    }
}

/*int main()
{
    using namespace cukd::common;
    int k = 30;
    float  cutOffRadius = std::numeric_limits<float>::infinity();

    int numPoints = 200000;
    float3* d_points = generatePoints(numPoints);

    int numQueries = 100000;
    float3* d_queries = generatePoints(numQueries);

    std::vector<float> points_pos_x; points_pos_x.resize(numPoints);
    std::vector<float> points_pos_y; points_pos_y.resize(numPoints);
    std::vector<float> points_pos_z; points_pos_z.resize(numPoints);

    for (int i = 0; i < numPoints; i++)
    {
        points_pos_x[i] = d_points[i].x;
        points_pos_y[i] = d_points[i].y;
        points_pos_z[i] = d_points[i].z;
    }

    std::vector<float> queries_pos_x; queries_pos_x.resize(numQueries);
    std::vector<float> queries_pos_y; queries_pos_y.resize(numQueries);
    std::vector<float> queries_pos_z; queries_pos_z.resize(numQueries);

    for (int i = 0; i < numQueries; i++)
    {
        queries_pos_x[i] = d_queries[i].x;
        queries_pos_y[i] = d_queries[i].y;
        queries_pos_z[i] = d_queries[i].z;
    }

    std::vector<float> new_points_pos_x; new_points_pos_x.resize(numPoints);
    std::vector<float> new_points_pos_y; new_points_pos_y.resize(numPoints);
    std::vector<float> new_points_pos_z; new_points_pos_z.resize(numPoints);

    std::vector<float> bound_upper;
    bound_upper.resize(3);
    std::vector<float> bound_lower;
    bound_lower.resize(3);

    GenerateKDTreeForGWN(points_pos_x, points_pos_y, points_pos_z,
             new_points_pos_x, new_points_pos_y, new_points_pos_z,
             bound_upper, bound_lower);

    std::vector<std::vector<int>> results_pos_index; results_pos_index.resize(numQueries);
    //std::vector<std::vector<float>> results_pos_y; results_pos_y.resize(numQueries);
    //std::vector<std::vector<float>> results_pos_z; results_pos_z.resize(numQueries);

    for (int i = 0; i < numQueries; i++)
    {
        std::vector<int> c_result_pos_index; c_result_pos_index.resize(k);
        //std::vector<float> c_result_pos_y; c_result_pos_y.resize(k);
        //std::vector<float> c_result_pos_z; c_result_pos_z.resize(k);

        results_pos_index[i] = c_result_pos_index;
        //results_pos_y[i] = c_result_pos_y;
        //results_pos_z[i] = c_result_pos_z;
    }

    CudaKNNWithKDTreeforGWN(new_points_pos_x, new_points_pos_y, new_points_pos_z,
        queries_pos_x, queries_pos_y, queries_pos_z,
        bound_upper, bound_lower,
        results_pos_index,
        k
    );

    
    /*for (int i = 0; i < numQueries; i++)
    {
        std::cout << i << " th:" << std::endl;
        std::cout << (float)d_queries[i].x << " " << (float)d_queries[i].y << " " << (float)d_queries[i].z << std::endl;
        for (int u = 0; u < k; u++)
        {
            int index = d_results[i * k + u];
            std::cout << (float)d_points[index].x << " " << (float)d_points[index].y << " " << (float)d_points[index].z << std::endl;
            std::cout << (float)d_points_c[index].x << " " << (float)d_points_c[index].y << " " << (float)d_points_c[index].z << std::endl;
        }

        std::cout << std::endl;
    }
}*/