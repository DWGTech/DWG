#include "hip/hip_runtime.h"
# include "TurboDWG.h"

#define gpuErrchk(ans) { gpuAssert((ans), #ans, __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* command, const char* file, int line, bool abort = true)
{
	//printf("%s = %i\n", command, (int)code);
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

TurboDWG::TurboDWG(Eigen::MatrixXf& octree_leaves, Eigen::MatrixXf& leaves_Ns, Eigen::VectorXf& leaves_As, Eigen::VectorXf radii, Eigen::MatrixXf& grid_points, int res_x, int res_y, int res_z, std::vector<float> bound_upper_kdtree, std::vector<float> bound_lower_kdtree)
{
	int point_cloud_size = octree_leaves.rows();
	hipMallocManaged((char**)&this->m_Point_Cloud_GPU, point_cloud_size * sizeof(*this->m_Point_Cloud_GPU));
	std::unique_ptr<float3[]> point_cloud_cpu = std::make_unique<float3[]>(size_t(point_cloud_size));
	for (int i = 0; i < point_cloud_size; i++)
	{
		point_cloud_cpu[i] = make_float3(octree_leaves(i, 0), octree_leaves(i, 1), octree_leaves(i, 2));
	}
	gpuErrchk(hipMemcpy(this->m_Point_Cloud_GPU, point_cloud_cpu.get(), point_cloud_size * sizeof(float3), hipMemcpyHostToDevice));
	this->m_Point_Cloud_size = point_cloud_size;

	int leaves_Ns_size = leaves_Ns.rows();
	hipMallocManaged((char**)&this->m_Normals_GPU, leaves_Ns_size * sizeof(*this->m_Normals_GPU));
	std::unique_ptr<float3[]> leaves_Ns_cpu = std::make_unique<float3[]>(size_t(leaves_Ns_size));
	for (int i = 0; i < leaves_Ns_size; i++)
	{
		leaves_Ns_cpu[i] = make_float3(leaves_Ns(i, 0), leaves_Ns(i, 1), leaves_Ns(i, 2));
	}
	gpuErrchk(hipMemcpy(this->m_Normals_GPU, leaves_Ns_cpu.get(), leaves_Ns_size * sizeof(float3), hipMemcpyHostToDevice));

	int As_size = leaves_As.size();
	//gpuErrchk(hipMalloc(&i_As_gpu, i_As_size * sizeof(float)));
	hipMallocManaged((char**)&this->m_As_GPU, As_size * sizeof(*this->m_As_GPU));
	std::unique_ptr<float[]> As_cpu = std::make_unique<float[]>(size_t(As_size));
	for (int i = 0; i < As_size; i++)
	{
		As_cpu[i] = leaves_As[i];
	}
	gpuErrchk(hipMemcpy(this->m_As_GPU, As_cpu.get(), As_size * sizeof(float), hipMemcpyHostToDevice));

	int radii_size = radii.size();
	//gpuErrchk(hipMalloc(&i_As_gpu, i_As_size * sizeof(float)));
	hipMallocManaged((char**)&this->m_Radii_GPU, radii_size * sizeof(*this->m_Radii_GPU));
	std::unique_ptr<float[]> radii_cpu = std::make_unique<float[]>(size_t(radii_size));
	for (int i = 0; i < radii_size; i++)
	{
		radii_cpu[i] = radii[i];
	}
	gpuErrchk(hipMemcpy(this->m_Radii_GPU, radii_cpu.get(), radii_size * sizeof(float), hipMemcpyHostToDevice));

	int grid_points_size = grid_points.rows();
	hipMallocManaged((char**)&this->m_Grid_Points_GPU, grid_points_size * sizeof(*this->m_Grid_Points_GPU));
	std::unique_ptr<float3[]> grid_points_cpu = std::make_unique<float3[]>(size_t(grid_points_size));
	for (int i = 0; i < grid_points_size; i++)
	{
		grid_points_cpu[i] = make_float3(grid_points(i, 0), grid_points(i, 1), grid_points(i, 2));
	}
	gpuErrchk(hipMemcpy(this->m_Grid_Points_GPU, grid_points_cpu.get(), grid_points_size * sizeof(float3), hipMemcpyHostToDevice));
	this->m_Grid_Points_size = grid_points_size;

	this->m_Grid_Res_X = res_x;
	this->m_Grid_Res_Y = res_y;
	this->m_Grid_Res_Z = res_z;

	hipMallocManaged((char**)&this->m_PC_WN_GPU, point_cloud_size * sizeof(*this->m_PC_WN_GPU));
	hipMallocManaged((char**)&this->m_GP_WN_GPU, grid_points_size * sizeof(*this->m_GP_WN_GPU));

	hipMallocManaged((char**)&this->m_Bound_Upper_KDT_GPU, sizeof(*this->m_Bound_Upper_KDT_GPU));
	this->m_Bound_Upper_KDT_GPU[0].x = bound_upper_kdtree[0];
	this->m_Bound_Upper_KDT_GPU[0].y = bound_upper_kdtree[1];
	this->m_Bound_Upper_KDT_GPU[0].z = bound_upper_kdtree[2];
	hipMallocManaged((char**)&this->m_Bound_Lower_KDT_GPU, sizeof(*this->m_Bound_Lower_KDT_GPU));
	this->m_Bound_Lower_KDT_GPU[0].x = bound_lower_kdtree[0];
	this->m_Bound_Lower_KDT_GPU[0].y = bound_lower_kdtree[1];
	this->m_Bound_Lower_KDT_GPU[0].z = bound_lower_kdtree[2];

	std::cout << "Res X: " << this->m_Grid_Res_X << " " << "Res Y: " << this->m_Grid_Res_Y << " " << "Res Z: " << this->m_Grid_Res_Z << std::endl;
	std::cout << "GPU Point Cloud Size: " << this->m_Point_Cloud_size << std::endl;
}

void TurboDWG::PreTurboDWG(int point_cloud_size, Eigen::VectorXf& leaves_As, Eigen::MatrixXf& grid_points, int res_x, int res_y, int res_z)
{
	int As_size = point_cloud_size;
	//gpuErrchk(hipMalloc(&i_As_gpu, i_As_size * sizeof(float)));
	hipMallocManaged((char**)&this->m_As_GPU, As_size * sizeof(*this->m_As_GPU));
	std::unique_ptr<float[]> As_cpu = std::make_unique<float[]>(size_t(As_size));
	for (int i = 0; i < As_size; i++)
	{
		As_cpu[i] = leaves_As[i];
	}
	gpuErrchk(hipMemcpy(this->m_As_GPU, As_cpu.get(), As_size * sizeof(float), hipMemcpyHostToDevice));

	int radii_size = point_cloud_size;
	//gpuErrchk(hipMalloc(&i_As_gpu, i_As_size * sizeof(float)));
	hipMallocManaged((char**)&this->m_Radii_GPU, radii_size * sizeof(*this->m_Radii_GPU));

	int grid_points_size = grid_points.rows();
	hipMallocManaged((char**)&this->m_Grid_Points_GPU, grid_points_size * sizeof(*this->m_Grid_Points_GPU));
	std::unique_ptr<float3[]> grid_points_cpu = std::make_unique<float3[]>(size_t(grid_points_size));
	for (int i = 0; i < grid_points_size; i++)
	{
		grid_points_cpu[i] = make_float3(grid_points(i, 0), grid_points(i, 1), grid_points(i, 2));
	}
	gpuErrchk(hipMemcpy(this->m_Grid_Points_GPU, grid_points_cpu.get(), grid_points_size * sizeof(float3), hipMemcpyHostToDevice));
	this->m_Grid_Points_size = grid_points_size;

	this->m_Grid_Res_X = res_x;
	this->m_Grid_Res_Y = res_y;
	this->m_Grid_Res_Z = res_z;

	hipMallocManaged((char**)&this->m_PC_WN_GPU, point_cloud_size * sizeof(*this->m_PC_WN_GPU));
	hipMallocManaged((char**)&this->m_GP_WN_GPU, grid_points_size * sizeof(*this->m_GP_WN_GPU));

	std::cout << "GPU Point Cloud Size: " << point_cloud_size << std::endl;
	std::cout << "Res X: " << this->m_Grid_Res_X << " " << "Res Y: " << this->m_Grid_Res_Y << " " << "Res Z: " << this->m_Grid_Res_Z << std::endl;
}

__global__
void AddWindNumbers(float* __restrict__ winding_numbers,
	float* __restrict__ winding_sum,
	int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		atomicAdd(winding_sum, winding_numbers[tid]);
	}
}

__global__
void ResetZero(float3* __restrict__ vector,
	int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		vector[tid] = make_float3(0.0f, 0.0f, 0.0f);
	}
}

__global__
void LoadZeroData(float3* __restrict__ zero_points,
	float3* __restrict__ nonzero_points,
	int* __restrict__ zero_points_index,
	int* __restrict__ nonzero_points_index,
	float3* __restrict__ nonzero_points_normal,
	float* __restrict__ tmp_lengths,
	float3* __restrict__ point_pos,
	float3* __restrict__ point_normal,
	int* c_zero_index,
	int* c_nonzero_index,
	int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		if (tmp_lengths[tid] < 1e-10)
		{
			int c_ato_index = atomicAdd(c_zero_index, 1);

			zero_points[c_ato_index] = point_pos[tid];
			zero_points_index[c_ato_index] = tid;
		}
		else
		{
			int c_ato_index = atomicAdd(c_nonzero_index, 1);

			nonzero_points[c_ato_index] = point_pos[tid];
			nonzero_points_index[c_ato_index] = tid;
			nonzero_points_normal[c_ato_index] = point_normal[tid];
		}
	}
}

__global__
void UpdateNormals(float3* __restrict__ prev_normals,
	int* __restrict__ knn_indices,
	float3* __restrict__ normals,
	int knn_size, int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		for (int j = 0; j < knn_size; j++)
		{
			int idx = knn_indices[tid * knn_size + j];

			/*if (idx == 666)
			{
				printf("prev_normals[idx].x: %f\n", prev_normals[idx].x);
				printf("normals[tid].x: %f\n", normals[tid].x);
			}*/
			/*printf("idx: %d\n", idx);*/

			atomicAdd(&(prev_normals[idx].x), normals[tid].x);
			atomicAdd(&(prev_normals[idx].y), normals[tid].y);
			atomicAdd(&(prev_normals[idx].z), normals[tid].z);
		}
	}
}

__global__
void LoadNormals(float3* __restrict__ prev_normals,
	int* __restrict__ knn_indices,
	float3* __restrict__ normals,
	int knn_size, int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		float3 sum_normal = make_float3(0.0f, 0.0f, 0.0f);

		for (int j = 0; j < knn_size; j++)
		{
			int idx = knn_indices[tid * knn_size + j];
			sum_normal.x += prev_normals[idx].x;
			sum_normal.y += prev_normals[idx].y;
			sum_normal.z += prev_normals[idx].z;
		}
		sum_normal = make_float3(sum_normal.x / ((float)knn_size), sum_normal.y / ((float)knn_size), sum_normal.z / ((float)knn_size));

		float length = __fsqrt_rn(sum_normal.x * sum_normal.x +
			sum_normal.y * sum_normal.y +
			sum_normal.z * sum_normal.z);

		float normal_scale = __frcp_rn(length + 1e-8);

		normals[tid] = make_float3(sum_normal.x * normal_scale,
			sum_normal.y * normal_scale,
			sum_normal.z * normal_scale);
	}
}

__global__
void NormalizeNormals(float3* __restrict__ tmp_normals, float3* __restrict__ ori_normals, float* __restrict__ lengths, int* __restrict__ zero_size, int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		float length = __fsqrt_rn(tmp_normals[tid].x * tmp_normals[tid].x +
			tmp_normals[tid].y * tmp_normals[tid].y +
			tmp_normals[tid].z * tmp_normals[tid].z);
		lengths[tid] = length;

		float normal_scale = __frcp_rn(length + 1e-8);

		/*normals[tid] = make_float3(normals[tid].x * normal_scale,
								   normals[tid].y * normal_scale,
								   normals[tid].z * normal_scale);*/

		if (length < 1e-10)
		{
			atomicAdd(zero_size, 1);
		}
		else
		{
			ori_normals[tid] = make_float3(tmp_normals[tid].x * normal_scale,
				tmp_normals[tid].y * normal_scale,
				tmp_normals[tid].z * normal_scale);
		}
	}
}

__inline__ __device__
float3 NormalAdd(const float3& a, const float3& b) {
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__
void UpdateNonzeroNormals(
	int* __restrict__ knn_indices,
	float3* __restrict__ normals,
	float3* __restrict__ result,
	int com_size)
{
	int bid = blockIdx.x;

	if (bid >= com_size)
		return;

	__shared__ float3 shared_normals[10];
	int neighbor_start = 10 * bid;

	if (threadIdx.x < 10)
	{
		shared_normals[threadIdx.x] = normals[knn_indices[neighbor_start + threadIdx.x]];
	}
	__syncthreads();

	if (threadIdx.x < 10)
	{
		for (int stride = 1; stride < 10; stride *= 2)
		{
			int index = 2 * stride * threadIdx.x;
			if (index < 10 && (index + stride < 10))
			{
				shared_normals[index] = NormalAdd(shared_normals[index], shared_normals[index + stride]);
			}
			__syncthreads();
		}

		if (threadIdx.x == 0)
		{
			float length = __fsqrt_rn(shared_normals[0].x * shared_normals[0].x +
				shared_normals[0].y * shared_normals[0].y +
				shared_normals[0].z * shared_normals[0].z);

			float normal_scale = __frcp_rn(length + 1e-8);
			shared_normals[0] = make_float3(shared_normals[0].x * normal_scale,
				shared_normals[0].y * normal_scale,
				shared_normals[0].z * normal_scale);

			result[bid] = shared_normals[0];
		}
	}
}

__global__
void SimpleUpdateNonzeroNormals(
	int* __restrict__ knn_indices,
	int* __restrict__ zero_points_index,
	float3* __restrict__ nonzero_points_normal,
	float3* __restrict__ ori_normals,
	int k_size,
	int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		float3 sum_normal = make_float3(0.0f, 0.0f, 0.0f);

		for (int i = 0; i < k_size; i++)
		{
			int c_index = knn_indices[tid * k_size + i];
			sum_normal.x += nonzero_points_normal[c_index].x;
			sum_normal.y += nonzero_points_normal[c_index].y;
			sum_normal.z += nonzero_points_normal[c_index].z;

			/*printf("knn_indices[i]: %d", knn_indices[i]);*/
		}

		float length = __fsqrt_rn(sum_normal.x * sum_normal.x +
			sum_normal.y * sum_normal.y +
			sum_normal.z * sum_normal.z);

		float normal_scale = __frcp_rn(length + 1e-8);

		int ori_index = zero_points_index[tid];
		ori_normals[ori_index] = make_float3(sum_normal.x * normal_scale,
			sum_normal.y * normal_scale,
			sum_normal.z * normal_scale);

		//printf("ori_normals[ori_index]: (%f, %f, %f)\n", ori_normals[ori_index].x, ori_normals[ori_index].y, ori_normals[ori_index].z);
	}
}

__global__
void SimpleUpdateRadii(
	int* __restrict__ knn_indices,
	float3* __restrict__ points,
	float* __restrict__ radii,
	int k_size,
	int com_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < com_size)
	{
		float radii_value = 0;
		for (int i = 0; i < k_size; i++)
		{
			float3 point_diff;
			int c_index = knn_indices[tid * k_size + i];
			point_diff.x += (points[tid].x - points[c_index].x);
			point_diff.y += (points[tid].y - points[c_index].y);
			point_diff.z += (points[tid].z - points[c_index].z);

			radii_value += (point_diff.x * point_diff.x + point_diff.y * point_diff.y + point_diff.z * point_diff.z);
		}

		radii_value /= ((float)k_size);
		radii_value = __fsqrt_rn(radii_value);
		radii_value = (radii_value < 0.0015f) ? 0.0015f : radii_value;
		radii_value = (radii_value > 0.015f) ? 0.015f : radii_value;

		radii[tid] = radii_value;
	}
}

/*__global__
void UpdateNonzeroNormals_15(
	int* __restrict__ knn_indices,
	float3* __restrict__ normals,
	float3* __restrict__ result,
	int com_size)
{
	int bid = blockIdx.x;

	if (bid >= com_size)
		return;

	__shared__ float3 shared_normals[15];
	int neighbor_start = 15 * bid;

	if (threadIdx.x < 15)
	{
		shared_normals[threadIdx.x] = normals[knn_indices[neighbor_start + threadIdx.x]];
	}
	__syncthreads();

	if (threadIdx.x < 15)
	{
		for (int stride = 1; stride < 15; stride *= 2)
		{
			int index = 2 * stride * threadIdx.x;
			if (index < 15 && (index + stride < 15))
			{
				shared_normals[index] = NormalAdd(shared_normals[index], shared_normals[index + stride]);
			}
			__syncthreads();
		}

		if (threadIdx.x == 0)
		{
			float length = __fsqrt_rn(shared_normals[0].x * shared_normals[0].x +
				shared_normals[0].y * shared_normals[0].y +
				shared_normals[0].z * shared_normals[0].z);

			float normal_scale = __frcp_rn(length + 1e-8);
			shared_normals[0] = make_float3(shared_normals[0].x * normal_scale,
				shared_normals[0].y * normal_scale,
				shared_normals[0].z * normal_scale);

			result[bid] = shared_normals[0];
		}
	}
}*/

__global__
void AsFiltertoZero(
	float* __restrict__ As,
	int* __restrict__ As_indices,
	int filter_num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < filter_num)
	{
		int c_index = As_indices[tid];
		As[c_index] /= 1000.0f;
	}
}

clock_t TurboDWG::TurboScreenedDWGLaunch(std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC, float beta, float sigma_in, int total_iters, bool use_radii, float As_threshold, int input_normal, clock_t start_o, std::string mesh_path, int test_iter)
{
	hipMallocManaged((char**)&this->m_GWN_GPU, sizeof(*this->m_GWN_GPU));
	this->m_GWN_GPU->PreLibiGWN(O_PI, O_CH, O_CM, O_R, O_EC);

	float* winding_sum = 0;
	hipMallocManaged((char**)&winding_sum, sizeof(*winding_sum));

	float3* mid_points = 0;
	float3* mid_normals = 0;

	int mesh_buffer_size = std::max((this->m_Grid_Res_X * this->m_Grid_Res_Y * this->m_Grid_Res_Z) / 2, 1000000);
	float3* mid_points_buffer = 0;
	float3* mid_normals_buffer = 0;
	hipMallocManaged((char**)&mid_points_buffer, mesh_buffer_size * sizeof(*mid_points_buffer));
	hipMallocManaged((char**)&mid_normals_buffer, mesh_buffer_size * sizeof(*mid_normals_buffer));

	int* mid_points_knn_gpu = 0;
	int* zero_points_knn_gpu = 0;

	float3* tmp_normals = 0;
	hipMallocManaged((char**)&tmp_normals, this->m_Point_Cloud_size * sizeof(*tmp_normals));
	float* tmp_lengths = 0;
	hipMallocManaged((char**)&tmp_lengths, this->m_Point_Cloud_size * sizeof(*tmp_lengths));
	int* zero_size = 0;
	hipMallocManaged((char**)&zero_size, sizeof(*zero_size));

	int* c_zero_index = 0;
	hipMallocManaged((char**)&c_zero_index, sizeof(*c_zero_index));
	int* c_nonzero_index = 0;
	hipMallocManaged((char**)&c_nonzero_index, sizeof(*c_nonzero_index));

	float3* zero_points = 0;
	float3* nonzero_points = 0;
	int* zero_points_index = 0;
	int* nonzero_points_index = 0;
	float3* nonzero_points_normal = 0;

	float3* nonzero_points_upper = 0;
	float3* nonzero_points_lower = 0;
	hipMallocManaged((char**)&nonzero_points_upper, sizeof(*nonzero_points_upper));
	hipMallocManaged((char**)&nonzero_points_lower, sizeof(*nonzero_points_lower));

	float3* tmp_zero_normals = 0;

	clock_t end_o = clock();
	std::cout << "Prepare Time = " << (end_o - start_o) / CLOCKS_PER_SEC << std::endl;

	if (input_normal == 3)
	{
		TurboNormalInitialization turbo_normal_init;
		turbo_normal_init.GaussMapInit(this->m_Point_Cloud_GPU, this->m_Normals_GPU, this->m_Bound_Upper_KDT_GPU, this->m_Bound_Lower_KDT_GPU, this->m_Point_Cloud_size);

		std::cout << "Gauss Map Normal Initialized" << std::endl;
	}

	OutlierFilter(this->m_As_GPU, As_threshold, this->m_Point_Cloud_size);

	clock_t start_iter = clock();
	int c_iter = 0;
	while (c_iter < total_iters)
	{
		clock_t start, finish;
		start = clock();

		std::cout << "iter = " << c_iter << std::endl;

		clock_t start_gwn, finish_gwn;
		start_gwn = clock();

		float sigma = sigma_in;
		if (c_iter < 4) {
			sigma = 20.0f; // 20.0

		}
		if (c_iter > 0 && c_iter % 7 == 0) {
			sigma = 10.0f; // 10.0
		}

		this->m_GWN_GPU->UpdateMassNormals(this->m_Normals_GPU, this->m_As_GPU);
		this->m_GWN_GPU->TurboLibiScreenedGWNLaunch(this->m_Point_Cloud_GPU, this->m_Point_Cloud_GPU, this->m_Point_Cloud_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_PC_WN_GPU, use_radii);
		this->m_GWN_GPU->TurboLibiScreenedGWNLaunch(this->m_Grid_Points_GPU, this->m_Point_Cloud_GPU, this->m_Grid_Points_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_GP_WN_GPU, use_radii);
		finish_gwn = clock();
		std::cout << "GWN time = " << (finish_gwn - start_gwn) / CLOCKS_PER_SEC << std::endl;

		thrust::device_ptr<float> pc_wn_thrust_ptr(this->m_PC_WN_GPU);
		float pc_wn_sum = thrust::reduce(pc_wn_thrust_ptr, pc_wn_thrust_ptr + this->m_Point_Cloud_size, 0.0f, thrust::plus<float>());
		float mean_wn = pc_wn_sum / ((float)this->m_Point_Cloud_size);

		std::cout << "W_mean = " << mean_wn << std::endl;

		if (c_iter % 5 == 0 && c_iter != 0)
		{
			mean_wn *= 0.5;
		}

		TurboMC turbomc;
		std::string mesh_full_path = "nothing";
		int num_tri = turbomc.TurboMarchingCubes(this->m_GP_WN_GPU, this->m_Grid_Points_GPU, this->m_Grid_Res_X, this->m_Grid_Res_Y, this->m_Grid_Res_Z, mean_wn, c_iter, mesh_path, mesh_full_path, mid_points_buffer, mid_normals_buffer, test_iter);
		hipMallocManaged((char**)&mid_points, num_tri * sizeof(*mid_points));
		hipMallocManaged((char**)&mid_normals, num_tri * sizeof(*mid_normals));
		hipMemcpy(mid_points, mid_points_buffer, num_tri * sizeof(*mid_points_buffer), hipMemcpyDeviceToDevice);
		hipMemcpy(mid_normals, mid_normals_buffer, num_tri * sizeof(*mid_normals_buffer), hipMemcpyDeviceToDevice);

		int k_size = 15;
		hipMallocManaged((void**)&mid_points_knn_gpu, num_tri * k_size * sizeof(*mid_points_knn_gpu));
		CudaKNNWithKDTreeforDWG(this->m_Point_Cloud_GPU,
			this->m_Point_Cloud_size,
			mid_points,
			num_tri,
			this->m_Bound_Upper_KDT_GPU, this->m_Bound_Lower_KDT_GPU,
			mid_points_knn_gpu,
			k_size
		);

		int threadsPerBlock_normalize = 128;
		int blocksPerGrid_normalize = (this->m_Point_Cloud_size + threadsPerBlock_normalize - 1) / threadsPerBlock_normalize;
		ResetZero << < blocksPerGrid_normalize, threadsPerBlock_normalize >> > (tmp_normals, this->m_Point_Cloud_size);
		hipDeviceSynchronize();

		int threadsPerBlock = 128;
		int blocksPerGrid = (num_tri + threadsPerBlock - 1) / threadsPerBlock;
		UpdateNormals << < blocksPerGrid, threadsPerBlock >> > (tmp_normals, mid_points_knn_gpu, mid_normals, k_size, num_tri);
		hipDeviceSynchronize();

		zero_size[0] = 0;
		NormalizeNormals << < blocksPerGrid_normalize, threadsPerBlock_normalize >> > (tmp_normals, this->m_Normals_GPU, tmp_lengths, zero_size, this->m_Point_Cloud_size);
		hipDeviceSynchronize();

		int zero_k_size = 10;
		if (zero_size[0] > zero_k_size)
		{
			int nonzero_size = this->m_Point_Cloud_size - zero_size[0];
			hipMallocManaged((char**)&zero_points, zero_size[0] * sizeof(*zero_points));
			hipMallocManaged((char**)&nonzero_points, nonzero_size * sizeof(*nonzero_points));
			hipMallocManaged((char**)&zero_points_index, zero_size[0] * sizeof(*zero_points_index));
			hipMallocManaged((char**)&nonzero_points_index, nonzero_size * sizeof(*nonzero_points_index));
			hipMallocManaged((char**)&nonzero_points_normal, nonzero_size * sizeof(*nonzero_points_normal));

			c_zero_index[0] = 0;
			c_nonzero_index[0] = 0;
			LoadZeroData << < blocksPerGrid_normalize, threadsPerBlock_normalize >> > (zero_points, nonzero_points, zero_points_index, nonzero_points_index, nonzero_points_normal, tmp_lengths,
				this->m_Point_Cloud_GPU, this->m_Normals_GPU, c_zero_index, c_nonzero_index, this->m_Point_Cloud_size);
			hipDeviceSynchronize();

			GenerateKDTreeForDWG(nonzero_points, nonzero_points_normal, nonzero_size, nonzero_points_upper, nonzero_points_lower);

			hipMallocManaged((void**)&zero_points_knn_gpu, zero_size[0] * zero_k_size * sizeof(*zero_points_knn_gpu));
			CudaKNNWithKDTreeforDWG(nonzero_points,
				nonzero_size,
				zero_points,
				zero_size[0],
				nonzero_points_upper, nonzero_points_lower,
				zero_points_knn_gpu,
				zero_k_size
			);

			std::cout << "zero_size[0] = " << zero_size[0] << std::endl;
			hipMallocManaged((void**)&tmp_zero_normals, zero_size[0] * sizeof(*tmp_zero_normals));
			//int threadsPerBlock_zero = 32;
			//int blocksPerGrid_zero = (zero_size[0] + threadsPerBlock_zero - 1) / threadsPerBlock_zero;
			//UpdateNonzeroNormals << < blocksPerGrid_zero, threadsPerBlock_zero >> > (zero_points_knn_gpu, this->m_Normals_GPU, tmp_zero_normals, zero_size[0]);

			int threadsPerBlock_zero = 128;
			int blocksPerGrid_zero = (zero_size[0] + threadsPerBlock - 1) / threadsPerBlock;
			SimpleUpdateNonzeroNormals << < blocksPerGrid_zero, threadsPerBlock_zero >> > (zero_points_knn_gpu, zero_points_index, nonzero_points_normal, this->m_Normals_GPU, zero_k_size, zero_size[0]);
			hipDeviceSynchronize();
			/*for (int i = 0; i < zero_size[0]; i++)
			{
				this->m_Normals_GPU[zero_points_index[i]] = tmp_zero_normals[i];
			}*/

			std::cout << "number of zero normals = " << zero_size[0] << std::endl;

			hipFree(zero_points_knn_gpu);
			zero_points_knn_gpu = 0;
			hipFree(zero_points);
			zero_points = 0;
			hipFree(nonzero_points);
			nonzero_points = 0;
			hipFree(zero_points_index);
			zero_points_index = 0;
			hipFree(nonzero_points_index);
			nonzero_points_index = 0;
			hipFree(nonzero_points_normal);
			nonzero_points_normal = 0;
			hipFree(tmp_zero_normals);
			tmp_zero_normals = 0;
		}

		hipFree(mid_points);
		mid_points = 0;
		hipFree(mid_normals);
		mid_normals = 0;
		hipFree(mid_points_knn_gpu);
		mid_points_knn_gpu = 0;

		c_iter++;

		finish = clock();
		std::cout << "iter time = " << (finish - start) / CLOCKS_PER_SEC << std::endl;
		std::cout << std::endl;
	}

	hipFree(winding_sum); winding_sum = 0;
	hipFree(tmp_normals); tmp_normals = 0;
	hipFree(tmp_lengths); tmp_lengths = 0;
	hipFree(zero_size); zero_size = 0;
	hipFree(nonzero_points_upper); nonzero_points_upper = 0;
	hipFree(nonzero_points_lower); nonzero_points_lower = 0;
	hipFree(this->m_Grid_Points_GPU); this->m_Grid_Points_GPU = 0;
	hipFree(this->m_GP_WN_GPU); this->m_GP_WN_GPU = 0;

	clock_t end_iter = clock();

	return end_iter - start_iter;
}

void TurboDWG::UpdateRadii(int k_size)
{
	int* knn_points = 0;
	hipMallocManaged((void**)&knn_points, this->m_Point_Cloud_size * k_size * sizeof(*knn_points));
	CudaKNNWithKDTreeforDWG(this->m_Point_Cloud_GPU,
		this->m_Point_Cloud_size,
		this->m_Point_Cloud_GPU,
		this->m_Point_Cloud_size,
		this->m_Bound_Upper_KDT_GPU, this->m_Bound_Lower_KDT_GPU,
		knn_points,
		k_size
	);

	int threadsPerBlock = 128;
	int blocksPerGrid = (this->m_Point_Cloud_size + threadsPerBlock - 1) / threadsPerBlock;
	SimpleUpdateRadii << < blocksPerGrid, threadsPerBlock >> > (knn_points, this->m_Point_Cloud_GPU, this->m_Radii_GPU, k_size, this->m_Point_Cloud_size);
	hipDeviceSynchronize();
}

void TurboDWG::UpdateAs(Eigen::VectorXd& leaves_As)
{
	std::unique_ptr<float[]> h_As = std::make_unique<float[]>(size_t(this->m_Point_Cloud_size));
	for (int i = 0; i < this->m_Point_Cloud_size; i++)
	{
		h_As[i] = (float)leaves_As[i];
	}
	gpuErrchk(hipMemcpy(this->m_As_GPU, h_As.get(), this->m_Point_Cloud_size * sizeof(float), hipMemcpyHostToDevice));
}

void TurboDWG::PreGeneration(Eigen::MatrixXf& cloud_points, Eigen::VectorXf& points_As, Eigen::MatrixXf& grid_points, int res_x, int res_y, int res_z, std::vector<float> bound_upper_kdtree, std::vector<float> bound_lower_kdtree, int k_normal_size, Eigen::MatrixXd& out_normal)
{
	hipFree(this->m_PC_WN_GPU); this->m_PC_WN_GPU = 0;
	hipFree(this->m_As_GPU); this->m_As_GPU = 0;
	hipFree(this->m_Radii_GPU); this->m_Radii_GPU = 0;

	this->m_GWN_GPU->EarseMemory();

	PreTurboDWG(cloud_points.rows(), points_As, grid_points, res_x, res_y, res_z);

	float3* cloud_points_gpu = 0;
	int cloud_points_size = cloud_points.rows();
	hipMallocManaged((char**)&cloud_points_gpu, cloud_points_size * sizeof(*cloud_points_gpu));
	std::unique_ptr<float3[]> cloud_points_cpu = std::make_unique<float3[]>(size_t(cloud_points_size));
	for (int i = 0; i < cloud_points_size; i++)
	{
		cloud_points_cpu[i] = make_float3(cloud_points(i, 0), cloud_points(i, 1), cloud_points(i, 2));
	}
	gpuErrchk(hipMemcpy(cloud_points_gpu, cloud_points_cpu.get(), cloud_points_size * sizeof(float3), hipMemcpyHostToDevice));

	int* knn_normal_points = 0;
	//int k_normal_size = 1; //DO NOT change it
	hipMallocManaged((void**)&knn_normal_points, cloud_points_size * k_normal_size * sizeof(*knn_normal_points));
	CudaKNNWithKDTreeforDWG(this->m_Point_Cloud_GPU,
		this->m_Point_Cloud_size,
		cloud_points_gpu,
		cloud_points_size,
		this->m_Bound_Upper_KDT_GPU, this->m_Bound_Lower_KDT_GPU,
		knn_normal_points,
		k_normal_size
	);

	hipFree(this->m_Point_Cloud_GPU); this->m_Point_Cloud_GPU = 0;
	float3* points_normal_gpu = 0;
	hipMallocManaged((char**)&points_normal_gpu, cloud_points_size * sizeof(*points_normal_gpu));

	int threadsPerBlock = 128;
	int blocksPerGrid = (cloud_points_size + threadsPerBlock - 1) / threadsPerBlock;
	LoadNormals << < blocksPerGrid, threadsPerBlock >> > (this->m_Normals_GPU, knn_normal_points, points_normal_gpu, k_normal_size, cloud_points_size);
	hipDeviceSynchronize();

	hipFree(this->m_Normals_GPU); this->m_Normals_GPU = 0;

	this->m_Bound_Upper_KDT_GPU[0].x = bound_upper_kdtree[0];
	this->m_Bound_Upper_KDT_GPU[0].y = bound_upper_kdtree[1];
	this->m_Bound_Upper_KDT_GPU[0].z = bound_upper_kdtree[2];

	this->m_Bound_Lower_KDT_GPU[0].x = bound_lower_kdtree[0];
	this->m_Bound_Lower_KDT_GPU[0].y = bound_lower_kdtree[1];
	this->m_Bound_Lower_KDT_GPU[0].z = bound_lower_kdtree[2];

	this->m_Point_Cloud_GPU = cloud_points_gpu;
	this->m_Point_Cloud_size = cloud_points_size;
	this->m_Normals_GPU = points_normal_gpu;

	std::unique_ptr<float3[]> h_normal = std::make_unique<float3[]>(size_t(this->m_Point_Cloud_size));
	hipMemcpy(h_normal.get(), this->m_Normals_GPU, this->m_Point_Cloud_size * sizeof(float3), hipMemcpyDeviceToHost);
	for (int i = 0; i < this->m_Point_Cloud_size; ++i)
	{
		out_normal(i, 0) = (double)h_normal[i].x;
		out_normal(i, 1) = (double)h_normal[i].y;
		out_normal(i, 2) = (double)h_normal[i].z;
	}
}

void TurboDWG::GenerateMesh(Eigen::MatrixXf& grid_points, int res_x, int res_y, int res_z, float beta, float sigma, int k_size, bool use_radii, std::string mesh_full_path, clock_t iter_time, clock_t start)
{
	clock_t start_time = clock();
	float3* grid_points_gpu = 0;
	int grid_points_size = grid_points.rows();
	hipMallocManaged((char**)&grid_points_gpu, grid_points_size * sizeof(*grid_points_gpu));
	std::unique_ptr<float3[]> grid_points_cpu = std::make_unique<float3[]>(size_t(grid_points_size));
	for (int i = 0; i < grid_points_size; i++)
	{
		grid_points_cpu[i] = make_float3(grid_points(i, 0), grid_points(i, 1), grid_points(i, 2));
	}
	gpuErrchk(hipMemcpy(grid_points_gpu, grid_points_cpu.get(), grid_points_size * sizeof(float3), hipMemcpyHostToDevice));

	float* grid_point_gwn = 0;
	hipMallocManaged((char**)&grid_point_gwn, grid_points_size * sizeof(*grid_point_gwn));

	UpdateRadii(k_size);
	this->m_GWN_GPU->UpdateMassNormals(this->m_Normals_GPU, this->m_As_GPU);
	this->m_GWN_GPU->TurboLibiScreenedGWNLaunch(this->m_Point_Cloud_GPU, this->m_Point_Cloud_GPU, this->m_Point_Cloud_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_PC_WN_GPU, use_radii);
	this->m_GWN_GPU->TurboLibiScreenedGWNLaunch(grid_points_gpu, this->m_Point_Cloud_GPU, grid_points_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, grid_point_gwn, use_radii);

	thrust::device_ptr<float> pc_wn_thrust_ptr(this->m_PC_WN_GPU);
	float pc_wn_sum = thrust::reduce(pc_wn_thrust_ptr, pc_wn_thrust_ptr + this->m_Point_Cloud_size, 0.0f, thrust::plus<float>());
	float mean_wn = pc_wn_sum / ((float)this->m_Point_Cloud_size);

	int mesh_buffer_size = std::max((res_x * res_y * res_z) / 2, 1000000);
	float3* mid_points_buffer = 0;
	float3* mid_normals_buffer = 0;
	hipMallocManaged((char**)&mid_points_buffer, mesh_buffer_size * sizeof(*mid_points_buffer));
	hipMallocManaged((char**)&mid_normals_buffer, mesh_buffer_size * sizeof(*mid_normals_buffer));

	TurboMC turbomc;
	std::string mesh_path = "nothing";
	int num_tri = turbomc.TurboMarchingCubes(grid_point_gwn, grid_points_gpu, res_x, res_y, res_z, mean_wn, 123456, mesh_path, mesh_full_path, mid_points_buffer, mid_normals_buffer, 999999);

	clock_t end_time = clock();
	std::cout << "Iter Time = " << (end_time - start_time + iter_time) / CLOCKS_PER_SEC << std::endl;
	std::cout << "Total Time = " << (end_time - start) / CLOCKS_PER_SEC << std::endl;
}

void TurboDWG::GenerateMesh(Eigen::MatrixXf& cloud_points, Eigen::VectorXf& points_As, Eigen::MatrixXf& grid_points,
	std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC,
	int res_x, int res_y, int res_z,
	std::vector<float> bound_upper_kdtree, std::vector<float> bound_lower_kdtree,
	float beta, float sigma, int k_size, int k_normal_size, bool use_radii, float As_threshold, std::string mesh_full_path, clock_t iter_time, clock_t start, int output_mode)
{
	clock_t start_time = clock();

	hipFree(this->m_PC_WN_GPU); this->m_PC_WN_GPU = 0;
	hipFree(this->m_As_GPU); this->m_As_GPU = 0;
	hipFree(this->m_Radii_GPU); this->m_Radii_GPU = 0;

	this->m_GWN_GPU->EarseMemory();

	PreTurboDWG(cloud_points.rows(), points_As, grid_points, res_x, res_y, res_z);

	float3* cloud_points_gpu = 0;
	int cloud_points_size = cloud_points.rows();
	hipMallocManaged((char**)&cloud_points_gpu, cloud_points_size * sizeof(*cloud_points_gpu));
	std::unique_ptr<float3[]> cloud_points_cpu = std::make_unique<float3[]>(size_t(cloud_points_size));
	for (int i = 0; i < cloud_points_size; i++)
	{
		cloud_points_cpu[i] = make_float3(cloud_points(i, 0), cloud_points(i, 1), cloud_points(i, 2));
	}
	gpuErrchk(hipMemcpy(cloud_points_gpu, cloud_points_cpu.get(), cloud_points_size * sizeof(float3), hipMemcpyHostToDevice));

	int* knn_normal_points = 0;
	//int k_normal_size = 1; //DO NOT change it
	hipMallocManaged((void**)&knn_normal_points, cloud_points_size * k_normal_size * sizeof(*knn_normal_points));
	CudaKNNWithKDTreeforDWG(this->m_Point_Cloud_GPU,
		this->m_Point_Cloud_size,
		cloud_points_gpu,
		cloud_points_size,
		this->m_Bound_Upper_KDT_GPU, this->m_Bound_Lower_KDT_GPU,
		knn_normal_points,
		k_normal_size
	);

	hipFree(this->m_Point_Cloud_GPU); this->m_Point_Cloud_GPU = 0;
	float3* points_normal_gpu = 0;
	hipMallocManaged((char**)&points_normal_gpu, cloud_points_size * sizeof(*points_normal_gpu));

	int threadsPerBlock = 128;
	int blocksPerGrid = (cloud_points_size + threadsPerBlock - 1) / threadsPerBlock;
	LoadNormals << < blocksPerGrid, threadsPerBlock >> > (this->m_Normals_GPU, knn_normal_points, points_normal_gpu, k_normal_size, cloud_points_size);
	hipDeviceSynchronize();

	hipFree(this->m_Normals_GPU); this->m_Normals_GPU = 0;

	this->m_Bound_Upper_KDT_GPU[0].x = bound_upper_kdtree[0];
	this->m_Bound_Upper_KDT_GPU[0].y = bound_upper_kdtree[1];
	this->m_Bound_Upper_KDT_GPU[0].z = bound_upper_kdtree[2];

	this->m_Bound_Lower_KDT_GPU[0].x = bound_lower_kdtree[0];
	this->m_Bound_Lower_KDT_GPU[0].y = bound_lower_kdtree[1];
	this->m_Bound_Lower_KDT_GPU[0].z = bound_lower_kdtree[2];

	this->m_Point_Cloud_GPU = cloud_points_gpu;
	this->m_Point_Cloud_size = cloud_points_size;
	this->m_Normals_GPU = points_normal_gpu;

	TurboLibiGWN turbogwn;
	turbogwn.PreLibiGWN(O_PI, O_CH, O_CM, O_R, O_EC);

	UpdateRadii(k_size);

	turbogwn.UpdateMassNormals(this->m_Normals_GPU, this->m_As_GPU);

	OutlierFilter(this->m_As_GPU, As_threshold, this->m_Point_Cloud_size);

	std::cout << "Start GWN Computation" << std::endl;
	turbogwn.TurboLibiScreenedGWNLaunch(this->m_Point_Cloud_GPU, this->m_Point_Cloud_GPU, this->m_Point_Cloud_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_PC_WN_GPU, use_radii);
	turbogwn.TurboLibiScreenedGWNLaunch(this->m_Grid_Points_GPU, this->m_Point_Cloud_GPU, this->m_Grid_Points_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_GP_WN_GPU, use_radii);

	thrust::device_ptr<float> pc_wn_thrust_ptr(this->m_PC_WN_GPU);
	float pc_wn_sum = thrust::reduce(pc_wn_thrust_ptr, pc_wn_thrust_ptr + this->m_Point_Cloud_size, 0.0f, thrust::plus<float>());
	float mean_wn = pc_wn_sum / ((float)this->m_Point_Cloud_size);

	int mesh_buffer_size = std::max((res_x * res_y * res_z) / 2, 1000000);
	float3* mid_points_buffer = 0;
	float3* mid_normals_buffer = 0;
	hipMallocManaged((char**)&mid_points_buffer, mesh_buffer_size * sizeof(*mid_points_buffer));
	hipMallocManaged((char**)&mid_normals_buffer, mesh_buffer_size * sizeof(*mid_normals_buffer));

	if (output_mode == 1 || output_mode == 3)
	{
		std::string mesh_path = "nothing";
		TurboMC turbomc;
		int num_tri = turbomc.TurboMarchingCubes(this->m_GP_WN_GPU, this->m_Grid_Points_GPU, res_x, res_y, res_z, mean_wn, 123456, mesh_path, mesh_full_path, mid_points_buffer, mid_normals_buffer, 999999);

		clock_t end_time = clock();
		std::cout << "Iter Time = " << iter_time / CLOCKS_PER_SEC << std::endl;
		std::cout << "Total Time = " << (end_time - start) / CLOCKS_PER_SEC << std::endl;
	}
	
	if(output_mode == 2 || output_mode == 3)
	{
		if (output_mode == 2)
		{
			clock_t end_time = clock();
			std::cout << "Iter Time = " << iter_time / CLOCKS_PER_SEC << std::endl;
			std::cout << "Total Time = " << (end_time - start) / CLOCKS_PER_SEC << std::endl;
		}

		std::cout << "Start save" << std::endl;

		std::ofstream outFile(mesh_full_path + ".xyz");
		for (int i = 0; i < this->m_Point_Cloud_size; i++)
		{
			outFile << this->m_Point_Cloud_GPU[i].x << " " << this->m_Point_Cloud_GPU[i].y << " " << this->m_Point_Cloud_GPU[i].z << " " <<
				this->m_Normals_GPU[i].x << " " << this->m_Normals_GPU[i].y << " " << this->m_Normals_GPU[i].z << "\n";
		}

		outFile.close();
		std::cout << "Finish save" << std::endl;
	}
}

void TurboDWG::MeshGeneration(std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC, int res_x, int res_y, int res_z, float beta, float sigma, int k_size, bool use_radii, float As_threshold, std::string mesh_full_path, clock_t start, std::chrono::high_resolution_clock::time_point start_time_s, int output_mode)
{
	TurboLibiGWN turbogwn;
	turbogwn.PreLibiGWN(O_PI, O_CH, O_CM, O_R, O_EC);

	turbogwn.UpdateGWNInfo(this->m_Normals_GPU, this->m_Point_Cloud_GPU, this->m_As_GPU);

	UpdateRadii(k_size);

	OutlierFilter(this->m_As_GPU, As_threshold, this->m_Point_Cloud_size);

	std::cout << "Start GWN Computation" << std::endl;
	turbogwn.TurboLibiScreenedGWNLaunch(this->m_Point_Cloud_GPU, this->m_Point_Cloud_GPU, this->m_Point_Cloud_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_PC_WN_GPU, use_radii);
	turbogwn.TurboLibiScreenedGWNLaunch(this->m_Grid_Points_GPU, this->m_Point_Cloud_GPU, this->m_Grid_Points_size, this->m_Normals_GPU, this->m_As_GPU, this->m_Radii_GPU, beta, sigma, this->m_GP_WN_GPU, use_radii);

	thrust::device_ptr<float> pc_wn_thrust_ptr(this->m_PC_WN_GPU);
	float pc_wn_sum = thrust::reduce(pc_wn_thrust_ptr, pc_wn_thrust_ptr + this->m_Point_Cloud_size, 0.0f, thrust::plus<float>());
	float mean_wn = pc_wn_sum / ((float)this->m_Point_Cloud_size);

	int mesh_buffer_size = std::max((res_x * res_y * res_z) / 2, 1000000);
	float3* c_mid_points_buffer = 0;
	float3* c_mid_normals_buffer = 0;
	hipMallocManaged((char**)&c_mid_points_buffer, mesh_buffer_size * sizeof(*c_mid_points_buffer));
	hipMallocManaged((char**)&c_mid_normals_buffer, mesh_buffer_size * sizeof(*c_mid_normals_buffer));

	if (output_mode == 1 || output_mode == 3)
	{
		std::string mesh_path = "nothing";
		TurboMC turbomc;
		int num_tri = turbomc.TurboMarchingCubes(this->m_GP_WN_GPU, this->m_Grid_Points_GPU, res_x, res_y, res_z, mean_wn, 123456, mesh_path, mesh_full_path, c_mid_points_buffer, c_mid_normals_buffer, 999999);

		clock_t end_time = clock();
		std::chrono::high_resolution_clock::time_point end_time_s = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time_s - start_time_s);

		std::cout << "Total Time = " << (end_time - start) / CLOCKS_PER_SEC << std::endl;
		std::cout << "Total Time (Wall time) = " << elapsed_time.count() << " s" << std::endl;
	}

	if (output_mode == 2 || output_mode == 3)
	{
		if (output_mode == 2)
		{
			clock_t end_time = clock();
			std::chrono::high_resolution_clock::time_point end_time_s = std::chrono::high_resolution_clock::now();
			std::chrono::duration<double> elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time_s - start_time_s);

			std::cout << "Total Time = " << (end_time - start) / CLOCKS_PER_SEC << std::endl;
			std::cout << "Total Time (Wall time) = " << elapsed_time.count() << " s" << std::endl;
		}

		std::cout << "Start save" << std::endl;

		std::ofstream outFile(mesh_full_path + ".xyz");
		for (int i = 0; i < this->m_Point_Cloud_size; i++)
		{
			outFile << this->m_Point_Cloud_GPU[i].x << " " << this->m_Point_Cloud_GPU[i].y << " " << this->m_Point_Cloud_GPU[i].z << " " <<
				this->m_Normals_GPU[i].x << " " << this->m_Normals_GPU[i].y << " " << this->m_Normals_GPU[i].z << "\n";
		}

		outFile.close();
		std::cout << "Finish save" << std::endl;
	}
}

void TurboDWG::OutlierFilter(float* As, float threshold, int point_num)
{
	thrust::device_vector<int> d_indices(point_num);
	thrust::sequence(d_indices.begin(), d_indices.end());

	thrust::device_vector<float> d_data(As, As + point_num);

	thrust::sort_by_key(d_data.begin(), d_data.end(), d_indices.begin(), thrust::greater<float>());

	int threshold_max_index = ceil((float)point_num * threshold);

	int threadsPerBlock = 128;
	int blocksPerGrid = (threshold_max_index + threadsPerBlock - 1) / threadsPerBlock;

	int* raw_indices = thrust::raw_pointer_cast(d_indices.data());

	AsFiltertoZero << < blocksPerGrid, threadsPerBlock >> > (As, raw_indices, threshold_max_index);
	hipDeviceSynchronize();
}