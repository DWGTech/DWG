#include "hip/hip_runtime.h"
#include "TurboNormalInit.h"

__global__
void GaussMapInitNormals(float3* __restrict__ point_pos,
	float3* __restrict__ point_normal,
	float3* __restrict__ bound_center,
	int point_num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < point_num)
	{
		float3 normal;
		normal.x = point_pos[tid].x - bound_center[0].x;
		normal.y = point_pos[tid].y - bound_center[0].y;
		normal.z = point_pos[tid].z - bound_center[0].z;

		float length = __fsqrt_rn(normal.x * normal.x +
			normal.y * normal.y +
			normal.z * normal.z);

		float normal_scale = __frcp_rn(length + 1e-8);

		point_normal[tid] = make_float3(normal.x * normal_scale,
			normal.y * normal_scale,
			normal.z * normal_scale);
	}
}

void TurboNormalInitialization::GaussMapInit(float3* point_pos, float3* point_normal, float3* bound_upper, float3* bound_lower, int point_num)
{
	float3* bound_center = 0;
	hipMallocManaged((char**)&bound_center, sizeof(*bound_center));

	bound_center[0].x = (bound_lower[0].x + bound_upper[0].x) / 2.0f;
	bound_center[0].y = (bound_lower[0].y + bound_upper[0].y) / 2.0f;
	bound_center[0].z = (bound_lower[0].z + bound_upper[0].z) / 2.0f;

	int threadsPerBlock = 128;
	int blocksPerGrid = (point_num + threadsPerBlock - 1) / threadsPerBlock;
	GaussMapInitNormals << < blocksPerGrid, threadsPerBlock >> > (point_pos, point_normal, bound_center, point_num);
	hipDeviceSynchronize();

	hipFree(bound_center); bound_center = 0;
}
