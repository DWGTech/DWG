#include "hip/hip_runtime.h"
#include "Turbo_GWN_l.h"

#define gpuErrchk(ans) { gpuAssert((ans), #ans, __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* command, const char* file, int line, bool abort = true)
{
	//printf("%s = %i\n", command, (int)code);
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

inline dim3 computeNbBlocks(unsigned int nbThreads, unsigned int nbThreadsPerBlock) {
	/*dim3 nbBlocks = ceil((float)(nbThreads) / (float)nbThreadsPerBlock);
	if (nbBlocks.x > 65535) {
		nbBlocks.y = ceil((float)nbBlocks.x / (float)65535);
		nbBlocks.x = 65535;
	}*/
	dim3 nbBlocks = ceil(((float)(nbThreads)+(float)nbThreadsPerBlock - 1.0f) / (float)nbThreadsPerBlock);
	return nbBlocks;
}

TurboLibiGWN::TurboLibiGWN(std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC, std::vector<geometrycentral::Vector3>& point_cloud)
{
	gpuErrchk(hipPeekAtLastError());

	unsigned int node_num = O_R.rows();

	this->m_Node_Size = node_num;
	hipMallocManaged((char**)&this->m_Octree_GPU, node_num * sizeof(*this->m_Octree_GPU));
	this->CopyToGPU(O_PI, O_CH, O_CM, O_R, O_EC, point_cloud);
}

TurboLibiGWN::TurboLibiGWN(std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC)
{
	gpuErrchk(hipPeekAtLastError());

	unsigned int node_num = O_R.rows();

	std::cout << "node_num: " << node_num << std::endl;

	this->m_Node_Size = node_num;
	hipMallocManaged((char**)&this->m_Octree_GPU, node_num * sizeof(*this->m_Octree_GPU));

	int child_num = 8;

	uint32_t num_indices_to_copy = 0;
	std::vector<unsigned int> whose_indices_to_copy;

	for (int i = 0; i < O_R.rows(); i++)
	{
		this->m_Octree_GPU[i].Mass_Center.x = O_CM(i, 0);
		this->m_Octree_GPU[i].Mass_Center.y = O_CM(i, 1);
		this->m_Octree_GPU[i].Mass_Center.z = O_CM(i, 2);

		this->m_Octree_GPU[i].Mass_Normal.x = O_EC(i, 0);
		this->m_Octree_GPU[i].Mass_Normal.y = O_EC(i, 1);
		this->m_Octree_GPU[i].Mass_Normal.z = O_EC(i, 2);

		this->m_Octree_GPU[i].Max_R = O_R[i];
	}

	this->m_Octree_GPU[0].Father = -1;
	for (int i = 0; i < O_CH.rows(); i++)
	{
		for (int j = 0; j < child_num; j++)
		{
			int c_ci = O_CH(i, j);

			if (c_ci != -1)
			{
				this->m_Octree_GPU[c_ci].Father = i;
				this->m_Octree_GPU[c_ci].IndexInFather = j;
			}
		}

		this->m_Octree_GPU[i].Point_Cloud_Num = O_PI[i].size();

		if (this->m_Octree_GPU[i].Point_Cloud_Num != 0)
		{
			whose_indices_to_copy.push_back(i);
			num_indices_to_copy += this->m_Octree_GPU[i].Point_Cloud_Num;
		}
	}

	if (num_indices_to_copy > 0)
	{
		unsigned int* point_indices = nullptr;
		hipMallocManaged((char**)&point_indices, num_indices_to_copy * sizeof(*point_indices));

		uint32_t offset = 0;
		for (unsigned int i : whose_indices_to_copy)
		{
			this->m_Octree_GPU[i].Point_Cloud_Indices = point_indices + offset;
			offset += this->m_Octree_GPU[i].Point_Cloud_Num;
		}

		std::unique_ptr<unsigned int[]> idxCPU = std::make_unique<unsigned int[]>(num_indices_to_copy * sizeof(unsigned int));
		unsigned int* idx = idxCPU.get();
		for (unsigned int i : whose_indices_to_copy)
		{
			for (unsigned int j = 0; j < this->m_Octree_GPU[i].Point_Cloud_Num; j++)
				*(idx++) = O_PI[i][j];
		}
		gpuErrchk(hipMemcpy(point_indices, idxCPU.get(), num_indices_to_copy * sizeof(unsigned int), hipMemcpyHostToDevice));
	}

	int* node_indices = nullptr;
	int total_node_num = ((int)O_CH.rows()) * child_num;
	hipMallocManaged((char**)&node_indices, total_node_num * sizeof(*node_indices));

	uint32_t offset = 0;
	for (int i = 0; i < O_CH.rows(); i++)
	{
		this->m_Octree_GPU[i].Children = node_indices + offset;
		offset += child_num;
	}

	std::unique_ptr<int[]> idxCPU = std::make_unique<int[]>(total_node_num * sizeof(int));
	int* idx = idxCPU.get();
	for (int i = 0; i < O_CH.rows(); i++)
	{
		for (int j = 0; j < 8; j++)
			*(idx++) = O_CH(i, j);
	}
	gpuErrchk(hipMemcpy(node_indices, idxCPU.get(), total_node_num * sizeof(int), hipMemcpyHostToDevice));

	std::cout << "node_num: " << node_num << std::endl;
}

void TurboLibiGWN::PreLibiGWN(std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC)
{
	gpuErrchk(hipPeekAtLastError());

	unsigned int node_num = O_R.rows();

	std::cout << "node_num: " << node_num << std::endl;

	this->m_Node_Size = node_num;
	hipMallocManaged((char**)&this->m_Octree_GPU, node_num * sizeof(*this->m_Octree_GPU));

	OctreeNodeLibiGPU* octree_cpu = new OctreeNodeLibiGPU[node_num];

	int child_num = 8;

	uint32_t num_indices_to_copy = 0;
	std::vector<unsigned int> whose_indices_to_copy;

	for (int i = 0; i < O_R.rows(); i++)
	{
		octree_cpu[i].Mass_Center.x = O_CM(i, 0);
		octree_cpu[i].Mass_Center.y = O_CM(i, 1);
		octree_cpu[i].Mass_Center.z = O_CM(i, 2);

		octree_cpu[i].Mass_Normal.x = O_EC(i, 0);
		octree_cpu[i].Mass_Normal.y = O_EC(i, 1);
		octree_cpu[i].Mass_Normal.z = O_EC(i, 2);

		octree_cpu[i].Max_R = O_R[i];
	}

	octree_cpu[0].Father = -1;
	for (int i = 0; i < O_CH.rows(); i++)
	{
		for (int j = 0; j < child_num; j++)
		{
			int c_ci = O_CH(i, j);

			if (c_ci != -1)
			{
				octree_cpu[c_ci].Father = i;
				octree_cpu[c_ci].IndexInFather = j;
			}
		}

		octree_cpu[i].Point_Cloud_Num = O_PI[i].size();

		if (octree_cpu[i].Point_Cloud_Num != 0)
		{
			whose_indices_to_copy.push_back(i);
			num_indices_to_copy += octree_cpu[i].Point_Cloud_Num;
		}
	}

	if (num_indices_to_copy > 0)
	{
		unsigned int* point_indices = nullptr;
		hipMallocManaged((char**)&point_indices, num_indices_to_copy * sizeof(*point_indices));

		uint32_t offset = 0;
		for (unsigned int i : whose_indices_to_copy)
		{
			octree_cpu[i].Point_Cloud_Indices = point_indices + offset;
			offset += octree_cpu[i].Point_Cloud_Num;
		}

		std::unique_ptr<unsigned int[]> idxCPU = std::make_unique<unsigned int[]>(num_indices_to_copy * sizeof(unsigned int));
		unsigned int* idx = idxCPU.get();
		for (unsigned int i : whose_indices_to_copy)
		{
			for (unsigned int j = 0; j < octree_cpu[i].Point_Cloud_Num; j++)
				*(idx++) = O_PI[i][j];
		}
		gpuErrchk(hipMemcpy(point_indices, idxCPU.get(), num_indices_to_copy * sizeof(unsigned int), hipMemcpyHostToDevice));
	}

	int* node_indices = nullptr;
	int total_node_num = ((int)O_CH.rows()) * child_num;
	hipMallocManaged((char**)&node_indices, total_node_num * sizeof(*node_indices));

	uint32_t offset = 0;
	for (int i = 0; i < O_CH.rows(); i++)
	{
		octree_cpu[i].Children = node_indices + offset;
		offset += child_num;
	}

	std::unique_ptr<int[]> idxCPU = std::make_unique<int[]>(total_node_num * sizeof(int));
	int* idx = idxCPU.get();
	for (int i = 0; i < O_CH.rows(); i++)
	{
		for (int j = 0; j < 8; j++)
			*(idx++) = O_CH(i, j);
	}
	gpuErrchk(hipMemcpy(node_indices, idxCPU.get(), total_node_num * sizeof(int), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(this->m_Octree_GPU, octree_cpu, node_num * sizeof(OctreeNodeLibiGPU), hipMemcpyHostToDevice));

	this->m_Octree_CPU = octree_cpu;
}

void TurboLibiGWN::CopyToGPU(std::vector<std::vector<int>>& O_PI, Eigen::MatrixXi& O_CH, Eigen::MatrixXd& O_CM, Eigen::VectorXd& O_R, Eigen::MatrixXd& O_EC, std::vector<geometrycentral::Vector3>& point_cloud)
{
	int child_num = 8;

	uint32_t num_indices_to_copy = 0;
	std::vector<unsigned int> whose_indices_to_copy;

	for (int i = 0; i < O_R.rows(); i++)
	{
		this->m_Octree_GPU[i].Mass_Center.x = O_CM(i, 0);
		this->m_Octree_GPU[i].Mass_Center.y = O_CM(i, 1);
		this->m_Octree_GPU[i].Mass_Center.z = O_CM(i, 2);

		this->m_Octree_GPU[i].Mass_Normal.x = O_EC(i, 0);
		this->m_Octree_GPU[i].Mass_Normal.y = O_EC(i, 1);
		this->m_Octree_GPU[i].Mass_Normal.z = O_EC(i, 2);

		this->m_Octree_GPU[i].Max_R = O_R[i];
	}

	this->m_Octree_GPU[0].Father = -1;
	for (int i = 0; i < O_CH.rows(); i++)
	{
		for (int j = 0; j < child_num; j++)
		{
			int c_ci = O_CH(i, j);

			if (c_ci != -1)
			{
				this->m_Octree_GPU[c_ci].Father = i;
				this->m_Octree_GPU[c_ci].IndexInFather = j;
			}
		}

		this->m_Octree_GPU[i].Point_Cloud_Num = O_PI[i].size();

		if (this->m_Octree_GPU[i].Point_Cloud_Num != 0)
		{
			whose_indices_to_copy.push_back(i);
			num_indices_to_copy += this->m_Octree_GPU[i].Point_Cloud_Num;
		}
	}

	if (num_indices_to_copy > 0)
	{
		unsigned int* point_indices = nullptr;
		hipMallocManaged((char**)&point_indices, num_indices_to_copy * sizeof(*point_indices));

		uint32_t offset = 0;
		for (unsigned int i : whose_indices_to_copy)
		{
			this->m_Octree_GPU[i].Point_Cloud_Indices = point_indices + offset;
			offset += this->m_Octree_GPU[i].Point_Cloud_Num;
		}

		std::unique_ptr<unsigned int[]> idxCPU = std::make_unique<unsigned int[]>(num_indices_to_copy * sizeof(unsigned int));
		unsigned int* idx = idxCPU.get();
		for (unsigned int i : whose_indices_to_copy)
		{
			for (unsigned int j = 0; j < this->m_Octree_GPU[i].Point_Cloud_Num; j++)
				*(idx++) = O_PI[i][j];
		}
		gpuErrchk(hipMemcpy(point_indices, idxCPU.get(), num_indices_to_copy * sizeof(unsigned int), hipMemcpyHostToDevice));
	}

	int* node_indices = nullptr;
	int total_node_num = ((int)O_CH.rows()) * child_num;
	hipMallocManaged((char**)&node_indices, total_node_num * sizeof(*node_indices));

	uint32_t offset = 0;
	for (int i = 0; i < O_CH.rows(); i++)
	{
		this->m_Octree_GPU[i].Children = node_indices + offset;
		offset += child_num;
	}

	std::unique_ptr<int[]> idxCPU = std::make_unique<int[]>(total_node_num * sizeof(int));
	int* idx = idxCPU.get();
	for (int i = 0; i < O_CH.rows(); i++)
	{
		for (int j = 0; j < 8; j++)
			*(idx++) = O_CH(i, j);
	}
	gpuErrchk(hipMemcpy(node_indices, idxCPU.get(), total_node_num * sizeof(int), hipMemcpyHostToDevice));

	int point_cloud_size = point_cloud.size();
	hipMallocManaged((char**)&this->m_Point_Cloud_GPU, point_cloud_size * sizeof(*this->m_Point_Cloud_GPU));
	std::unique_ptr<float3[]> point_cloud_cpu = std::make_unique<float3[]>(size_t(point_cloud_size));
	for (int i = 0; i < point_cloud_size; i++)
	{
		point_cloud_cpu[i] = make_float3(point_cloud[i].x, point_cloud[i].y, point_cloud[i].z);
	}
	gpuErrchk(hipMemcpy(this->m_Point_Cloud_GPU, point_cloud_cpu.get(), point_cloud_size * sizeof(float3), hipMemcpyHostToDevice));
}

__device__
float GWNLibiPStoQ(OctreeNodeLibiGPU* root_node, float3& q, float3* i_points, float3* i_Normals, float* i_As, float beta)
{
	float gwn = 0.0;
	bool continue_traversal = true;

	int iter_num = 0;
	int points_num = 0;

	int traversal_index = 0;

	int current_node_index = 0;

	while (continue_traversal)
	{
		if (root_node[current_node_index].Children[0] == -1)
		{
			for (int i_point_in_cn = 0; i_point_in_cn < root_node[current_node_index].Point_Cloud_Num; i_point_in_cn++)
			{
				int p_idx = root_node[current_node_index].Point_Cloud_Indices[i_point_in_cn];
				float3& p = i_points[p_idx];
				float3& N = i_Normals[p_idx];
				float _A = i_As[p_idx];

				float3 _R;
				_R.x = p.x - q.x; _R.y = p.y - q.y; _R.z = p.z - q.z;
				float _R_norm = sqrtf(_R.x * _R.x + _R.y * _R.y + _R.z * _R.z);
				float dot_RN = _R.x * N.x + _R.y * N.y + _R.z * N.z;
				dot_RN *= _A;
				dot_RN /= (4.0 * 3.14159265358979323 * _R_norm * _R_norm * _R_norm + 1e-8);
				gwn += dot_RN;

				//out_gwn[points_num] = gwn;

				points_num++;
			}

			while (true)
			{
				bool is_last_child = true;
				//int next_node = -1;
				if (root_node[current_node_index].IndexInFather == 7)
					is_last_child = true;
				else if (root_node[current_node_index].Father == -1)
				{
					continue_traversal = false;
					break;
				}
				else
				{
					traversal_index = root_node[current_node_index].IndexInFather + 1;
					current_node_index = root_node[current_node_index].Father;
					break;
				}
				if (root_node[current_node_index].Father != -1 && is_last_child)
				{
					current_node_index = root_node[current_node_index].Father;

					if (root_node[current_node_index].Father == -1)
					{
						continue_traversal = false;
						break;
					}
				}
				else
				{
					break;
				}
			}

		}
		else
		{
			//if (traversal_index == 0)
				//out_indices[iter_num] = current_node->Breadth_First_Index;

			bool descend = false;
			for (int c = traversal_index; c < 8; c++)
			{
				int cn_child = root_node[current_node_index].Children[c];
				if (root_node[cn_child].Point_Cloud_Num > 0)
				{
					float3& _massCenter = root_node[cn_child].Mass_Center;
					float _max_R = root_node[cn_child].Max_R;
					float3 MCToq;
					MCToq.x = _massCenter.x - q.x; MCToq.y = _massCenter.y - q.y; MCToq.z = _massCenter.z - q.z;
					float mctop_norm = sqrtf(MCToq.x * MCToq.x + MCToq.y * MCToq.y + MCToq.z * MCToq.z);

					if (mctop_norm > beta * _max_R)
					{
						if (root_node[cn_child].Children[0] == -1)
						{
							for (int i_point_in_cnc = 0; i_point_in_cnc < root_node[cn_child].Point_Cloud_Num; i_point_in_cnc++)
							{
								int p_idx = root_node[cn_child].Point_Cloud_Indices[i_point_in_cnc];
								float3& p = i_points[p_idx];
								float3& N = i_Normals[p_idx];
								float _A = i_As[p_idx];

								float3 _R;
								_R.x = p.x - q.x; _R.y = p.y - q.y; _R.z = p.z - q.z;
								float _R_norm = sqrtf(_R.x * _R.x + _R.y * _R.y + _R.z * _R.z);
								float dot_RN = _R.x * N.x + _R.y * N.y + _R.z * N.z;
								dot_RN *= _A;
								dot_RN /= (4.0 * 3.14159265358979323 * _R_norm * _R_norm * _R_norm + 1e-8);
								gwn += dot_RN;

								//out_gwn[points_num] = gwn;

								points_num++;
							}
						}
						else
						{
							float3& _massNormal = root_node[cn_child].Mass_Normal;

							//float3 _R;
							//_R.x = _massCenter.x - q.x; _R.y = _massCenter.y - q.y; _R.z = _massCenter.z - q.z;
							//float _R_norm = sqrtf(_R.x * _R.x + _R.y * _R.y + _R.z * _R.z);
							//float dot_RN = _R.x * _massNormal.x + _R.y * _massNormal.y + _R.z * _massNormal.z;
							float dot_RN = MCToq.x * _massNormal.x + MCToq.y * _massNormal.y + MCToq.z * _massNormal.z;

							//dot_RN /= (4.0 * 3.14159265358979323 * _R_norm * _R_norm * _R_norm + 1e-8);
							dot_RN /= (4.0 * 3.14159265358979323 * mctop_norm * mctop_norm * mctop_norm + 1e-8);
							gwn += dot_RN;

							//out_gwn[points_num] = gwn;

							points_num++;
						}
					}
					else
					{
						current_node_index = cn_child;
						descend = true;
						break;
					}
				}
			}

			traversal_index = 0;

			if (!descend)
			{
				while (true)
				{
					bool is_last_child = true;
					//int next_node = -1;
					if (root_node[current_node_index].IndexInFather == 7)
						is_last_child = true;
					else if (root_node[current_node_index].Father == -1)
					{
						continue_traversal = false;
						break;
					}
					else
					{
						traversal_index = root_node[current_node_index].IndexInFather + 1;
						current_node_index = root_node[current_node_index].Father;
						break;
					}
					if (root_node[current_node_index].Father != -1 && is_last_child)
					{
						current_node_index = root_node[current_node_index].Father;

						if (root_node[current_node_index].Father == -1)
						{
							continue_traversal = false;
							break;
						}
					}
					else
					{
						break;
					}
				}
			}
		}

		iter_num++;
	}

	return gwn;
}

__device__
float ScreenedGWNLibiPStoQ(OctreeNodeLibiGPU* __restrict__ root_node, float3& q, float3* __restrict__ i_points, float3* __restrict__ i_Normals, float* __restrict__ i_As, float* __restrict__ i_radii, float beta, float sigma, bool use_radii)
{
	float gwn = 0.0;
	bool continue_traversal = true;

	int iter_num = 0;
	int points_num = 0;

	int traversal_index = 0;

	int current_node_index = 0;

	float PI_4 = 4.0f * 3.14159265358979323;

	while (continue_traversal)
	{
		if (root_node[current_node_index].Children[0] == -1)
		{
			//out_indices[iter_num] = current_node->Breadth_First_Index;
			for (int i_point_in_cn = 0; i_point_in_cn < root_node[current_node_index].Point_Cloud_Num; i_point_in_cn++)
			{
				int p_idx = root_node[current_node_index].Point_Cloud_Indices[i_point_in_cn];
				float3& p = i_points[p_idx];
				float3& N = i_Normals[p_idx];
				float _A = i_As[p_idx];

				float3 _R;
				_R.x = p.x - q.x; _R.y = p.y - q.y; _R.z = p.z - q.z;
				float _R_norm = sqrtf(_R.x * _R.x + _R.y * _R.y + _R.z * _R.z);

				if (_R_norm > 1e-10)
				{
					float dot_RN = _R.x * N.x + _R.y * N.y + _R.z * N.z;
					dot_RN *= _A;

					float screened = (use_radii) ? i_radii[p_idx] : 0.0015f;
					bool sign = (dot_RN > 0.0f) ? true : false;

					if (screened > _R_norm)
					{
						float up = (screened * sigma + 1.0f) * dot_RN;
						float down = expf(screened * sigma) * (PI_4 * screened * screened * screened) + 1e-7;
						float res = up / down;

						if (sign && res < 0)
							gwn += (-1.0f * res);
						else if (!sign && res > 0)
						{
							gwn += (-1.0f * res);
						}
						else
						{
							gwn += res;
						}
					}
					else
					{
						float up = (_R_norm * sigma + 1.0f) * dot_RN;
						float down = expf(_R_norm * sigma) * (PI_4 * _R_norm * _R_norm * _R_norm) + 1e-7;
						float res = up / down;

						if (sign && res < 0)
							gwn += (-1.0f * res);
						else if (!sign && res > 0)
						{
							gwn += (-1.0f * res);
						}
						else
						{
							gwn += res;
						}
					}
				}

				//out_gwn[points_num] = gwn;

				points_num++;
			}

			while (true)
			{
				bool is_last_child = true;
				//int next_node = -1;
				if (root_node[current_node_index].IndexInFather == 7)
					is_last_child = true;
				else if (root_node[current_node_index].Father == -1)
				{
					continue_traversal = false;
					break;
				}
				else
				{
					traversal_index = root_node[current_node_index].IndexInFather + 1;
					current_node_index = root_node[current_node_index].Father;
					break;
				}
				if (root_node[current_node_index].Father != -1 && is_last_child)
				{
					current_node_index = root_node[current_node_index].Father;

					if (root_node[current_node_index].Father == -1)
					{
						continue_traversal = false;
						break;
					}
				}
				else
				{
					break;
				}
			}

		}
		else
		{
			//if (traversal_index == 0)
				//out_indices[iter_num] = current_node->Breadth_First_Index;

			bool descend = false;
			for (int c = traversal_index; c < 8; c++)
			{
				int cn_child = root_node[current_node_index].Children[c];
				if (root_node[cn_child].Point_Cloud_Num > 0)
				{
					float3& _massCenter = root_node[cn_child].Mass_Center;
					float _max_R = root_node[cn_child].Max_R;
					float3 MCToq;
					MCToq.x = _massCenter.x - q.x; MCToq.y = _massCenter.y - q.y; MCToq.z = _massCenter.z - q.z;
					float mctop_norm = sqrtf(MCToq.x * MCToq.x + MCToq.y * MCToq.y + MCToq.z * MCToq.z);

					if (mctop_norm > beta * _max_R)
					{
						if (root_node[cn_child].Children[0] == -1)
						{
							for (int i_point_in_cnc = 0; i_point_in_cnc < root_node[cn_child].Point_Cloud_Num; i_point_in_cnc++)
							{
								int p_idx = root_node[cn_child].Point_Cloud_Indices[i_point_in_cnc];
								float3& p = i_points[p_idx];
								float3& N = i_Normals[p_idx];
								float _A = i_As[p_idx];

								float3 _R;
								_R.x = p.x - q.x; _R.y = p.y - q.y; _R.z = p.z - q.z;
								float _R_norm = sqrtf(_R.x * _R.x + _R.y * _R.y + _R.z * _R.z);

								if (_R_norm > 1e-10)
								{
									float dot_RN = _R.x * N.x + _R.y * N.y + _R.z * N.z;
									dot_RN *= _A;

									float screened = (use_radii) ? i_radii[p_idx] : 0.0015f;
									bool sign = (dot_RN > 0.0f) ? true : false;

									if (screened > _R_norm)
									{
										float up = (screened * sigma + 1.0f) * dot_RN;
										float down = expf(screened * sigma) * (PI_4 * screened * screened * screened) + 1e-7;
										float res = up / down;

										if (sign && res < 0)
											gwn += (-1.0f * res);
										else if (!sign && res > 0)
										{
											gwn += (-1.0f * res);
										}
										else
										{
											gwn += res;
										}
									}
									else
									{
										float up = (_R_norm * sigma + 1.0f) * dot_RN;
										float down = expf(_R_norm * sigma) * (PI_4 * _R_norm * _R_norm * _R_norm) + 1e-7;
										float res = up / down;

										if (sign && res < 0)
											gwn += (-1.0f * res);
										else if (!sign && res > 0)
										{
											gwn += (-1.0f * res);
										}
										else
										{
											gwn += res;
										}
									}
								}

								points_num++;
							}
						}
						else
						{
							float3& _massNormal = root_node[cn_child].Mass_Normal;

							float _R_norm = mctop_norm;
							if (_R_norm > 1e-10)
							{
								float dot_RN = MCToq.x * _massNormal.x + MCToq.y * _massNormal.y + MCToq.z * _massNormal.z;

								float screened = 0.0015f;
								bool sign = (dot_RN > 0.0f) ? true : false;

								if (screened > _R_norm)
								{
									float up = (screened * sigma + 1.0f) * dot_RN;
									float down = expf(screened * sigma) * (PI_4 * screened * screened * screened) + 1e-7;
									float res = up / down;

									if (sign && res < 0)
										gwn += (-1.0f * res);
									else if (!sign && res > 0)
									{
										gwn += (-1.0f * res);
									}
									else
									{
										gwn += res;
									}
								}
								else
								{
									float up = (_R_norm * sigma + 1.0f) * dot_RN;
									float down = expf(_R_norm * sigma) * (PI_4 * _R_norm * _R_norm * _R_norm) + 1e-7;
									float res = up / down;

									if (sign && res < 0)
										gwn += (-1.0f * res);
									else if (!sign && res > 0)
									{
										gwn += (-1.0f * res);
									}
									else
									{
										gwn += res;
									}
								}
							}
							//out_gwn[points_num] = gwn;

							points_num++;
						}
					}
					else
					{
						current_node_index = cn_child;
						descend = true;
						break;
					}
				}
			}

			traversal_index = 0;

			if (!descend)
			{
				while (true)
				{
					bool is_last_child = true;
					//int next_node = -1;
					if (root_node[current_node_index].IndexInFather == 7)
						is_last_child = true;
					else if (root_node[current_node_index].Father == -1)
					{
						continue_traversal = false;
						break;
					}
					else
					{
						traversal_index = root_node[current_node_index].IndexInFather + 1;
						current_node_index = root_node[current_node_index].Father;
						break;
					}
					if (root_node[current_node_index].Father != -1 && is_last_child)
					{
						current_node_index = root_node[current_node_index].Father;

						if (root_node[current_node_index].Father == -1)
						{
							continue_traversal = false;
							break;
						}
					}
					else
					{
						break;
					}
				}
			}
		}

		iter_num++;
	}

	//printf("points_num: %d\n", points_num);

	return gwn;
}

__global__
void TurboLibiGWNKernal(int query_size, OctreeNodeLibiGPU* root_node, float3* q, float3* i_points, float3* i_Normals, float* i_As, float beta, float* gwn_results)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int tid_i = tid; tid_i < query_size; tid_i += stride)
	{
		gwn_results[tid_i] = GWNLibiPStoQ(root_node, q[tid_i], i_points, i_Normals, i_As, beta);
	}
}

__global__
void TurboLibiScreenedGWNKernal(int query_size, OctreeNodeLibiGPU* root_node, float3* q, float3* i_points, float3* i_Normals, float* i_As, float* i_radii, float beta, float sigma, float* gwn_results, bool use_radii)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int tid_i = tid; tid_i < query_size; tid_i += stride)
	{
		gwn_results[tid_i] = ScreenedGWNLibiPStoQ(root_node, q[tid_i], i_points, i_Normals, i_As, i_radii, beta, sigma, use_radii);
	}
}

void TurboLibiGWN::TurboLibiGWNLaunch(std::vector<geometrycentral::Vector3>& q, std::vector<geometrycentral::Vector3>& i_Normals, std::vector<float>& i_As, float& beta, float* out_gwn)
{
	float* gwn_results = 0;
	int query_size = q.size();
	hipMallocManaged((char**)&gwn_results, query_size * sizeof(*gwn_results));

	float3* q_gpu = 0;
	std::cout << "query_size: " << query_size << std::endl;
	hipMallocManaged((char**)&q_gpu, query_size * sizeof(*q_gpu));
	std::unique_ptr<float3[]> q_cpu = std::make_unique<float3[]>(size_t(query_size));
	for (int i = 0; i < query_size; i++)
	{
		q_cpu[i] = make_float3(q[i].x, q[i].y, q[i].z);
	}
	gpuErrchk(hipMemcpy(q_gpu, q_cpu.get(), query_size * sizeof(float3), hipMemcpyHostToDevice));

	float3* i_Normals_gpu = 0;
	int i_Normals_size = i_Normals.size();
	hipMallocManaged((char**)&i_Normals_gpu, i_Normals_size * sizeof(*i_Normals_gpu));
	std::unique_ptr<float3[]> i_Normals_cpu = std::make_unique<float3[]>(size_t(i_Normals_size));
	for (int i = 0; i < i_Normals_size; i++)
	{
		i_Normals_cpu[i] = make_float3(i_Normals[i].x, i_Normals[i].y, i_Normals[i].z);
	}
	gpuErrchk(hipMemcpy(i_Normals_gpu, i_Normals_cpu.get(), i_Normals_size * sizeof(float3), hipMemcpyHostToDevice));

	float* i_As_gpu = 0;
	int i_As_size = i_As.size();
	hipMallocManaged((char**)&i_As_gpu, i_As_size * sizeof(*i_As_gpu));
	std::unique_ptr<float[]> i_As_cpu = std::make_unique<float[]>(size_t(i_As_size));
	for (int i = 0; i < i_As_size; i++)
	{
		i_As_cpu[i] = i_As[i];
	}
	gpuErrchk(hipMemcpy(i_As_gpu, i_As_cpu.get(), i_As_size * sizeof(float), hipMemcpyHostToDevice));

	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks((unsigned int)query_size, blockSize);

	TurboLibiGWNKernal << <numBlocks, blockSize >> > ((int)query_size, this->m_Octree_GPU, q_gpu, this->m_Point_Cloud_GPU, i_Normals_gpu, i_As_gpu, beta, gwn_results);

	hipDeviceSynchronize();

	gpuErrchk(hipMemcpy(out_gwn, gwn_results, query_size * sizeof(float), hipMemcpyDeviceToHost));

	gpuErrchk(hipFree(gwn_results));
	gwn_results = nullptr;

	gpuErrchk(hipFree(q_gpu));
	q_gpu = nullptr;

	gpuErrchk(hipFree(i_Normals_gpu));
	i_Normals_gpu = nullptr;

	gpuErrchk(hipFree(i_As_gpu));
	i_As_gpu = nullptr;

	std::cout << "GWN Done" << std::endl;
}

void TurboLibiGWN::TurboLibiGWNLaunch(float3* query_points, float3* point_cloud, int query_size, float3* i_Normals, float* i_As, float& beta, float* out_gwn)
{
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks((unsigned int)query_size, blockSize);

	TurboLibiGWNKernal << <numBlocks, blockSize >> > (query_size, this->m_Octree_GPU, query_points, point_cloud, i_Normals, i_As, beta, out_gwn);

	hipDeviceSynchronize();

	std::cout << "out_gwn[666] " << out_gwn[666] << std::endl;

	std::cout << "GWN Done" << std::endl;
}

void TurboLibiGWN::TurboLibiScreenedGWNLaunch(float3* query_points, float3* point_cloud, int query_size, float3* i_Normals, float* i_As, float* i_radii, float& beta, float& sigma, float* out_gwn, bool use_radii)
{
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks((unsigned int)query_size, blockSize);

	TurboLibiScreenedGWNKernal << <numBlocks, blockSize >> > (query_size, this->m_Octree_GPU, query_points, point_cloud, i_Normals, i_As, i_radii, beta, sigma, out_gwn, use_radii);

	hipDeviceSynchronize();

	std::cout << "GWN Done" << std::endl;
}

__device__
float3 CalMassNormal(OctreeNodeLibiGPU* node, float3* i_Normals, float* i_As)
{
	float3 new_mass_normal;
	new_mass_normal.x = 0.0;
	new_mass_normal.y = 0.0;
	new_mass_normal.z = 0.0;
	for (int i = 0; i < node->Point_Cloud_Num; i++)
	{
		int curr_point_index = node->Point_Cloud_Indices[i];
		new_mass_normal.x += i_As[curr_point_index] * i_Normals[curr_point_index].x;
		new_mass_normal.y += i_As[curr_point_index] * i_Normals[curr_point_index].y;
		new_mass_normal.z += i_As[curr_point_index] * i_Normals[curr_point_index].z;
	}

	return new_mass_normal;
}

__global__
void UpdateMassNormalsKernal(int nodes_size, OctreeNodeLibiGPU* root_node, float3* i_Normals, float* i_As)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int tid_i = tid; tid_i < nodes_size; tid_i += stride)
	{
		root_node[tid_i].Mass_Normal = CalMassNormal(root_node + tid_i, i_Normals, i_As);
	}
}

__global__
void UpdateGWNInfoKernal(int nodes_size, OctreeNodeLibiGPU* root_node, float3* i_Normals, float3* point_cloud, float* i_As)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int tid_i = tid; tid_i < nodes_size; tid_i += stride)
	{
		OctreeNodeLibiGPU* node = root_node + tid_i;

		float3 new_mass_normal;
		new_mass_normal.x = 0.0;
		new_mass_normal.y = 0.0;
		new_mass_normal.z = 0.0;

		float3 new_mass_center;
		new_mass_center.x = 0.0;
		new_mass_center.y = 0.0;
		new_mass_center.z = 0.0;

		float areatotal = 0.0;

		for (int i = 0; i < node->Point_Cloud_Num; i++)
		{
			int curr_point_index = node->Point_Cloud_Indices[i];
			new_mass_normal.x += i_As[curr_point_index] * i_Normals[curr_point_index].x;
			new_mass_normal.y += i_As[curr_point_index] * i_Normals[curr_point_index].y;
			new_mass_normal.z += i_As[curr_point_index] * i_Normals[curr_point_index].z;

			areatotal += i_As[curr_point_index];
			new_mass_center.x += i_As[curr_point_index] * point_cloud[curr_point_index].x;
			new_mass_center.y += i_As[curr_point_index] * point_cloud[curr_point_index].y;
			new_mass_center.z += i_As[curr_point_index] * point_cloud[curr_point_index].z;
		}

		new_mass_center.x = new_mass_center.x / areatotal;
		new_mass_center.y = new_mass_center.y / areatotal;
		new_mass_center.z = new_mass_center.z / areatotal;

		float c_norm = 0;
		float max_norm = 0;
		for (int i = 0; i < node->Point_Cloud_Num; i++)
		{
			int curr_point_index = node->Point_Cloud_Indices[i];
			float3 point_diff;
			point_diff.x = point_cloud[curr_point_index].x - new_mass_center.x;
			point_diff.y = point_cloud[curr_point_index].y - new_mass_center.y;
			point_diff.z = point_cloud[curr_point_index].z - new_mass_center.z;

			c_norm = __fsqrt_rn(point_diff.x * point_diff.x +
				point_diff.y * point_diff.y +
				point_diff.z * point_diff.z);

			if (c_norm > max_norm)
			{
				max_norm = c_norm;
			}
		}

		root_node[tid_i].Mass_Normal = new_mass_normal;
		root_node[tid_i].Mass_Center = new_mass_center;
		root_node[tid_i].Max_R = max_norm;
	}
}

void TurboLibiGWN::UpdateMassNormals(std::vector<geometrycentral::Vector3>& i_Normals, std::vector<float>& i_As)
{
	float3* i_Normals_gpu = 0;
	int i_Normals_size = i_Normals.size();
	//gpuErrchk(hipMalloc(&i_Normals_gpu, i_Normals_size * sizeof(float3)));
	hipMallocManaged((char**)&i_Normals_gpu, i_Normals_size * sizeof(*i_Normals_gpu));
	std::unique_ptr<float3[]> i_Normals_cpu = std::make_unique<float3[]>(size_t(i_Normals_size));
	for (int i = 0; i < i_Normals_size; i++)
	{
		i_Normals_cpu[i] = make_float3(i_Normals[i].x, i_Normals[i].y, i_Normals[i].z);
	}
	gpuErrchk(hipMemcpy(i_Normals_gpu, i_Normals_cpu.get(), i_Normals_size * sizeof(float3), hipMemcpyHostToDevice));

	float* i_As_gpu = 0;
	int i_As_size = i_As.size();
	//gpuErrchk(hipMalloc(&i_As_gpu, i_As_size * sizeof(float)));
	hipMallocManaged((char**)&i_As_gpu, i_As_size * sizeof(*i_As_gpu));
	std::unique_ptr<float[]> i_As_cpu = std::make_unique<float[]>(size_t(i_As_size));
	for (int i = 0; i < i_As_size; i++)
	{
		i_As_cpu[i] = i_As[i];
	}
	gpuErrchk(hipMemcpy(i_As_gpu, i_As_cpu.get(), i_As_size * sizeof(float), hipMemcpyHostToDevice));

	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks((unsigned int)this->m_Node_Size, blockSize);

	UpdateMassNormalsKernal << <numBlocks, blockSize >> > (this->m_Node_Size, this->m_Octree_GPU, i_Normals_gpu, i_As_gpu);

	hipDeviceSynchronize();
}

void TurboLibiGWN::UpdateMassNormals(float3* i_Normals, float* i_As)
{
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks((unsigned int)this->m_Node_Size, blockSize);

	UpdateMassNormalsKernal << <numBlocks, blockSize >> > (this->m_Node_Size, this->m_Octree_GPU, i_Normals, i_As);

	hipDeviceSynchronize();
}

void TurboLibiGWN::UpdateGWNInfo(float3* i_Normals, float3* point_cloud, float* i_As)
{
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks((unsigned int)this->m_Node_Size, blockSize);

	UpdateGWNInfoKernal << <numBlocks, blockSize >> > (this->m_Node_Size, this->m_Octree_GPU, i_Normals, point_cloud, i_As);

	hipDeviceSynchronize();
}

void TurboLibiGWN::EarseMemory()
{
	for (int i = 0; i < m_Node_Size; i++)
	{
		if (m_Octree_CPU[i].Point_Cloud_Num > 0)
		{
			gpuErrchk(hipFree(m_Octree_CPU[i].Point_Cloud_Indices));
			break;
		}
	}

	gpuErrchk(hipFree(m_Octree_CPU[0].Children));

	delete[] m_Octree_CPU;
	m_Octree_CPU = nullptr;
	gpuErrchk(hipFree(m_Octree_GPU));
	m_Octree_GPU = nullptr;
}